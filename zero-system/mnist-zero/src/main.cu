#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>

#include <nn/mod.cuh>

struct Batch
{
	NdArray *x;
	NdArray *y;
};

std::vector<Batch> get_train_dataset(int batch_size)
{
	int img_row_cnt = 28;
	int img_col_cnt = 28;
	int img_area = img_row_cnt * img_col_cnt;
	int img_cnt = 60000;

	std::vector<Batch> batches;

	FILE *img_file = fopen("data/train-images.idx3-ubyte", "rb");
	FILE *lbl_file = fopen("data/train-labels.idx1-ubyte", "rb");

	fseek(img_file, sizeof(int) * 4, 0);
	unsigned char *img_buf = (unsigned char *)malloc((sizeof(unsigned char) * img_area * img_cnt));
	fread(img_buf, 1, (sizeof(unsigned char) * img_area * img_cnt), img_file);

	fseek(lbl_file, sizeof(int) * 2, 0);
	unsigned char *lbl_buf = (unsigned char *)malloc(sizeof(unsigned char) * img_cnt);
	fread(lbl_buf, 1, (sizeof(unsigned char) * img_cnt), lbl_file);

	fclose(img_file);
	fclose(lbl_file);

	float *img_flt_buf = (float *)malloc(sizeof(float) * (img_area * img_cnt));
	for (int i = 0; i < (img_area * img_cnt); i++)
	{
		img_flt_buf[i] = ((float)img_buf[i] / (255.0));
	}

	float *lbl_flt_buf = (float *)malloc(sizeof(float) * (img_cnt));
	for (int i = 0; i < (img_cnt); i++)
	{
		lbl_flt_buf[i] = ((float)lbl_buf[i]);
	}

	free(img_buf);
	free(lbl_buf);

	for (int i = 0; i < img_cnt / batch_size; i++)
	{
		auto x = NdArray::from_data(Shape(batch_size, 1, img_row_cnt, img_col_cnt), &img_flt_buf[i * batch_size * img_area]);
		auto y = NdArray::from_data(Shape(batch_size, 1), &lbl_flt_buf[i * batch_size]);
		auto oh_y = NdArray::one_hot(y, 9);
		delete y;

		batches.push_back({x, oh_y});
	}

	free(lbl_flt_buf);
	free(img_flt_buf);

	return batches;
}

Batch get_test_batch(int img_cnt)
{
	int img_row_cnt = 28;
	int img_col_cnt = 28;
	int img_area = img_row_cnt * img_col_cnt;

	FILE *img_file = fopen("data/t10k-images.idx3-ubyte", "rb");
	FILE *lbl_file = fopen("data/t10k-labels.idx1-ubyte", "rb");

	fseek(img_file, sizeof(int) * 4, 0);
	unsigned char *img_buf = (unsigned char *)malloc((sizeof(unsigned char) * img_area * img_cnt));
	fread(img_buf, 1, (sizeof(unsigned char) * img_area * img_cnt), img_file);

	fseek(lbl_file, sizeof(int) * 2, 0);
	unsigned char *lbl_buf = (unsigned char *)malloc(sizeof(unsigned char) * img_cnt);
	fread(lbl_buf, 1, (sizeof(unsigned char) * img_cnt), lbl_file);

	fclose(img_file);
	fclose(lbl_file);

	float *img_flt_buf = (float *)malloc(sizeof(float) * (img_area * img_cnt));
	for (int i = 0; i < (img_area * img_cnt); i++)
	{
		img_flt_buf[i] = ((float)img_buf[i] / (255.0));
	}

	float *lbl_flt_buf = (float *)malloc(sizeof(float) * (img_cnt));
	for (int i = 0; i < (img_cnt); i++)
	{
		lbl_flt_buf[i] = ((float)lbl_buf[i]);
	}

	free(img_buf);
	free(lbl_buf);

	auto x = NdArray::from_data(Shape(img_cnt, 1, img_row_cnt, img_col_cnt), img_flt_buf);
	auto y = NdArray::from_data(Shape(img_cnt, 1), lbl_flt_buf);
	auto oh_y = NdArray::one_hot(y, 9);
	delete y;

	free(lbl_flt_buf);
	free(img_flt_buf);

	return {x, oh_y};
}

void train_mnist(nn::Model *model, int batch_size, int epoch_cnt)
{
	auto train_ds = get_train_dataset(batch_size);

	int train_batch_cnt = train_ds.size();

	printf("EPOCH: %d\n", 0);

	for (int i = 0; i < epoch_cnt; i++)
	{
		for (int j = 0; j < train_batch_cnt; j++)
		{
			auto batch = &train_ds[j];
			auto x = batch->x;
			auto y = batch->y;

			auto p = model->forward(x);
			model->backward(p, y);
			model->step();
			delete p;

			if (_kbhit())
			{
				if (_getch() == 'q')
				{
					printf("Quitting...\n");
					return;
				}
			}
		}

		system("cls");
		printf("EPOCH: %d\n", i);
	}
}

void train_validate_mnist(nn::Model *model, int batch_size, int epoch_cnt, float validation_pct)
{
	auto train_ds = get_train_dataset(batch_size);

	int train_batch_cnt = train_ds.size();
	int validation_batch_cnt = (int)(train_batch_cnt * validation_pct);

	std::vector<int> validation_batch_idxs;
	for (int v = 0; v < validation_batch_cnt; v++)
	{
		validation_batch_idxs.push_back(rand() % train_batch_cnt);
	}

	for (int i = 0; i < epoch_cnt; i++)
	{
		float validation_loss = 0.0f;
		float validation_acc = 0.0f;

		for (int j = 0; j < train_batch_cnt; j++)
		{
			auto batch = &train_ds[j];
			auto x = batch->x;
			auto y = batch->y;

			bool validation_batch_flg = false;
			for (int v = 0; v < validation_batch_cnt; v++)
			{
				if (validation_batch_idxs[v] == j)
				{
					validation_batch_flg = true;
					break;
				}
			}

			if (validation_batch_flg)
			{
				auto p = model->forward(x);
				validation_loss += model->loss(p, y);
				validation_acc += model->accuracy(p, y);
				delete p;
			}
			else
			{
				auto p = model->forward(x);
				model->backward(p, y);
				model->step();
				delete p;
			}

			if (_kbhit())
			{
				if (_getch() == 'q')
				{
					printf("Quitting...\n");
					return;
				}
			}
		}

		printf("EPOCH: %d\tVALIDATION LOSS: %f\tVALIDATION ACCURACY: %f%%\n", i + 1,
			   (validation_loss / (float)validation_batch_cnt),
			   (validation_acc / (float)validation_batch_cnt) * 100.0f);
	}
}

void test_mnist(nn::Model *model)
{
	auto test_batch = get_test_batch(10000);
	auto x = test_batch.x;
	auto y = test_batch.y;

	model->change_batch_size(x->shape()[0]);

	auto p = model->forward(x);
	auto l = model->loss(p, y);
	printf("TEST LOSS: %f\tACCURACY: %f%%\n", l, model->accuracy(p, y) * 100.0f);

	delete p;

	delete x;
	delete y;
}

void check_grad(nn::Model *model)
{
	auto batch = get_test_batch(1);
	model->validate_gradients(batch.x, batch.y, true);
}

void grad_tests()
{
	auto m1 = new nn::Model();
	auto m2 = new nn::Model();
	auto m3 = new nn::Model();
	auto m4 = new nn::Model();

	int batch_size = 1;

	// m1
	{
		auto x = NdArray::random(true, Shape(batch_size, 64), 0.0f, 1.0f);
		auto y = NdArray::ones(true, Shape(batch_size, 1));

		m1->linear(x->shape(), 16);
		m1->tanh();
		m1->linear(16);
		m1->tanh();
		m1->linear(y->shape());
		m1->sigmoid();

		m1->set_loss(new nn::loss::MSE());
		m1->set_optimizer(new nn::optim::SGD(m1->parameters(), 0.01f));

		m1->summarize();
		m1->validate_gradients(x, y, false);

		delete x;
		delete y;
	}

	// m2
	{
		auto x = NdArray::random(true, Shape(batch_size, 64), 0.0f, 1.0f);
		auto y = NdArray::zeros(true, Shape(batch_size, 10));
		y->set_val(3, 1.0f);

		m2->linear(x->shape(), 16);
		m2->tanh();
		m2->linear(16);
		m2->tanh();
		m2->linear(y->shape());
		m2->sigmoid();

		m2->set_loss(new nn::loss::CrossEntropy());
		m2->set_optimizer(new nn::optim::SGD(m2->parameters(), 0.01f));

		m2->summarize();
		m2->validate_gradients(x, y, false);

		delete x;
		delete y;
	}

	// m3
	{
		auto x = NdArray::random(true, Shape(batch_size, 2, 16, 16), 0.0f, 1.0f);
		auto y = NdArray::zeros(true, Shape(batch_size, 4));
		y->set_val(3, 1.0f);

		m3->conv2d(x->shape(), Shape(4, 2, 2, 2), nn::layer::Padding{1, 1}, nn::layer::Stride{2, 2});
		m3->tanh();
		m3->conv2d(Shape(4, 4, 3, 3), nn::layer::Padding{1, 1}, nn::layer::Stride{1, 1});
		m3->tanh();
		m3->linear(16);
		m3->sigmoid();
		m3->linear(y->shape());
		m3->sigmoid();

		m3->set_loss(new nn::loss::MSE());
		m3->set_optimizer(new nn::optim::SGD(m3->parameters(), 0.01f));

		m3->summarize();
		m3->validate_gradients(x, y, false);

		delete x;
		delete y;
	}

	// m4
	{
		auto x = NdArray::random(true, Shape(batch_size, 2, 21, 14), 0.0f, 1.0f);
		auto y = NdArray::zeros(true, Shape(batch_size, 4));
		y->set_val(3, 1.0f);

		m4->conv2d(x->shape(), Shape(4, 2, 3, 2), nn::layer::Stride{3, 2});
		m4->tanh();
		m4->conv2d(Shape(4, 4, 2, 2), nn::layer::Stride{1, 1});
		m4->tanh();
		m4->linear(16);
		m4->sigmoid();
		m4->linear(y->shape());
		m4->sigmoid();

		m4->set_loss(new nn::loss::CrossEntropy());
		m4->set_optimizer(new nn::optim::SGD(m4->parameters(), 0.01f));

		m4->summarize();
		m4->validate_gradients(x, y, false);

		delete x;
		delete y;
	}

	delete m1;
	delete m2;
	delete m3;
	delete m4;
}

int main(int argc, char **argv)
{
	printf("MNIST-ZERO\n\n");
	srand(time(NULL));

	auto model = new nn::Model();
	int batch_size = 256;

	model->conv2d(Shape(batch_size, 1, 28, 28), Shape(16, 1, 3, 3), nn::layer::Padding{1, 1}, nn::layer::Stride{3, 3});
	model->relu();
	model->conv2d(Shape(8, 16, 3, 3), nn::layer::Padding{1, 1}, nn::layer::Stride{3, 3});
	model->relu();
	model->linear(128);
	model->relu();
	model->linear(64);
	model->relu();
	model->linear(Shape(batch_size, 10));
	model->sigmoid();

	model->set_loss(new nn::loss::CrossEntropy());
	model->set_optimizer(new nn::optim::SGD(model->parameters(), 1.0f));

	model->summarize();

	train_mnist(model, batch_size, 1000);
	// train_validate_mnist(model, batch_size, 1000, 0.10f);
	test_mnist(model);
	// check_grad(model);
	// grad_tests();

	return 0;
}