#include "hip/hip_runtime.h"
#include "ndarray.cuh"

__global__ void k_set_all(float *data, int cnt, float val)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        data[tid] = val;
    }
}

__global__ void k_pad(float *dst, float *src, int dst_row_cnt, int dst_col_cnt, int src_row_cnt, int src_col_cnt, int pad_row_cnt, int pad_col_cnt)
{
    int src_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int src_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (src_col_idx < src_col_cnt && src_row_idx < src_row_cnt)
    {
        dst[(src_row_idx + pad_row_cnt) * dst_col_cnt + (src_col_idx + pad_col_cnt)] = src[src_row_idx * src_col_cnt + src_col_idx];
    }
}

Shape::Shape()
{
}

Shape::Shape(int dim_1)
{
    this->dims_.push_back(dim_1);
}

Shape::Shape(int dim_1, int dim_2)
{
    this->dims_.push_back(dim_1);
    this->dims_.push_back(dim_2);
}

Shape::Shape(int dim_1, int dim_2, int dim_3)
{
    this->dims_.push_back(dim_1);
    this->dims_.push_back(dim_2);
    this->dims_.push_back(dim_3);
}

Shape::Shape(int dim_1, int dim_2, int dim_3, int dim_4)
{
    this->dims_.push_back(dim_1);
    this->dims_.push_back(dim_2);
    this->dims_.push_back(dim_3);
    this->dims_.push_back(dim_4);
}

Shape::Shape(std::vector<int> dims)
{
    this->dims_ = dims;
}

Shape::Shape(int dim_1, Shape shape)
{
    this->dims_.push_back(dim_1);
    for (int dim_i : shape.dims_)
    {
        this->dims_.push_back(dim_i);
    }
}

Shape::~Shape()
{
}

int Shape::operator[](int idx) const
{
    return this->dims_[idx];
}

bool Shape::operator==(const Shape &other)
{
    if (this->dims_.size() != other.dims_.size())
    {
        return false;
    }

    for (int i = 0; i < this->dims_.size(); i++)
    {
        if (this->dims_[i] != other.dims_[i])
        {
            return false;
        }
    }

    return true;
}

bool Shape::operator!=(const Shape &other)
{
    return !(*this == other);
}

void Shape::print()
{
    int cnt = this->num_dims();

    for (int i = 0; i < cnt; i++)
    {
        printf("%d", this->dims_[i]);

        if (i < cnt - 1)
        {
            printf("x");
        }
    }
}

std::vector<int> Shape::dims()
{
    return this->dims_;
}

int Shape::num_dims()
{
    return this->dims_.size();
}

int Shape::dims_size()
{
    int size = 1;

    for (int i = 0; i < this->num_dims(); i++)
    {
        size *= this->dims_[i];
    }

    return size;
}

NdArray::NdArray(NdArray &src)
{
    this->cuda_ = src.cuda_;
    this->shape_ = src.shape_;

    size_t size = this->size();

    if (src.cuda_)
    {
        hipMalloc(&this->data_, size);
        hipMemcpy(this->data_, src.data_, size, hipMemcpyDeviceToDevice);
    }
    else
    {
        this->data_ = (float *)malloc(size);
        memcpy(this->data_, src.data_, size);
    }
}

NdArray::NdArray(bool cuda, Shape shape)
{
    this->cuda_ = cuda;
    this->shape_ = shape;

    size_t size = this->size();

    if (cuda)
    {
        hipMalloc(&this->data_, size);
    }
    else
    {
        this->data_ = (float *)malloc(size);
    }
}

NdArray::~NdArray()
{
    if (this->cuda_)
    {
        hipFree(this->data_);
    }
    else
    {
        free(this->data_);
    }
}

NdArray *NdArray::from_data(Shape shape, float *data)
{
    NdArray *arr = new NdArray(false, shape);
    hipMemcpy(arr->data_, data, arr->size(), hipMemcpyDefault);
    return arr;
}

NdArray *NdArray::from_csv(const char *path)
{
    FILE *file_ptr = fopen(path, "rb");

    fseek(file_ptr, 0L, SEEK_END);
    long long file_size = FileUtils::get_file_size(path);
    rewind(file_ptr);

    char *buf = (char *)malloc(file_size + 1);
    memset(buf, 0, file_size + 1);
    fread(buf, 1, file_size, file_ptr);

    fclose(file_ptr);

    int buf_idx = 0;

    int row_cnt = 0;
    int col_cnt = 0;

    while (buf[buf_idx] != '\n')
    {
        if (buf[buf_idx] == ',')
        {
            col_cnt++;
        }

        buf_idx++;
    }

    col_cnt++;
    buf_idx++;

    int lst_row_idx = 0;
    for (int i = buf_idx; i < file_size; i++)
    {
        if (buf[i] == '\n')
        {
            row_cnt++;
            lst_row_idx = i;
        }
    }

    // If file does not end in newline, add to the row count.
    if (lst_row_idx < file_size - 1)
    {
        row_cnt++;
    }

    NdArray *arr = new NdArray(false, Shape(row_cnt, col_cnt));

    char temp_buf[64];
    memset(temp_buf, 0, 64);
    int temp_buf_idx = 0;
    int row_idx = 0;
    int col_idx = 0;

    for (; buf_idx < file_size; buf_idx++)
    {
        while (buf[buf_idx] != ',' && buf[buf_idx] != '\n' && buf_idx < file_size)
        {
            if (buf[buf_idx] != '"')
            {
                temp_buf[temp_buf_idx++] = buf[buf_idx];
            }

            buf_idx++;
        }

        if (buf[buf_idx] == ',')
        {
            arr->set_val(row_idx * col_cnt + col_idx, (float)atof(temp_buf));
            memset(temp_buf, 0, 64);
            col_idx++;
            temp_buf_idx = 0;
        }
        else if (buf[buf_idx] == '\n')
        {
            arr->set_val(row_idx * col_cnt + col_idx, (float)atof(temp_buf));
            memset(temp_buf, 0, 64);
            row_idx++;
            col_idx = 0;
            temp_buf_idx = 0;
        }
    }

    // Make sure to grab the last bit before we finish up!
    if (temp_buf_idx > 0)
    {
        arr->set_val(row_idx * col_cnt + col_idx, (float)atof(temp_buf));
        memset(temp_buf, 0, 64);
        row_idx++;
        col_idx = 0;
        temp_buf_idx = 0;
    }

    free(buf);

    return arr;
}

void NdArray::to_csv(const char *path, NdArray *arr)
{
    int dim_cnt = arr->num_dims();

    if (dim_cnt == 1)
    {
        int cnt = arr->shape_[0];

        FILE *file_ptr = fopen(path, "w");

        fprintf(file_ptr, "col\n");

        for (int i = 0; i < cnt; i++)
        {
            fprintf(file_ptr, "%f\n", arr->get_val(i));
        }

        fclose(file_ptr);
    }
    else if (dim_cnt == 2)
    {

        int row_cnt = arr->shape_[0];
        int col_cnt = arr->shape_[1];

        FILE *file_ptr = fopen(path, "w");

        for (int j = 0; j < col_cnt; j++)
        {

            if (j < col_cnt - 1)
            {
                fprintf(file_ptr, "col_%d,", j);
            }
            else
            {
                fprintf(file_ptr, "col_%d", j);
            }
        }
        fprintf(file_ptr, "\n");

        for (int i = 0; i < row_cnt; i++)
        {
            for (int j = 0; j < col_cnt; j++)
            {
                if (j < col_cnt - 1)
                {
                    fprintf(file_ptr, "%f,", arr->get_val(i * col_cnt + j));
                }
                else
                {
                    fprintf(file_ptr, "%f", arr->get_val(i * col_cnt + j));
                }
            }
            fprintf(file_ptr, "\n");
        }
        fclose(file_ptr);
    }
    else
    {
        return;
    }
}

void NdArray::to_file(const char *path, NdArray *arr)
{
    bool orig_cuda = arr->cuda_;

    FILE *file_ptr = fopen(path, "wb");

    arr->to_cpu();

    fwrite(arr->data_, sizeof(float), arr->count(), file_ptr);

    fclose(file_ptr);

    if (orig_cuda)
    {
        arr->to_cuda();
    }
}

NdArray *NdArray::zeros(bool cuda, Shape shape)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->zeros();

    return arr;
}

NdArray *NdArray::ones(bool cuda, Shape shape)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->ones();

    return arr;
}

NdArray *NdArray::full(bool cuda, Shape shape, float val)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->full(val);

    return arr;
}

NdArray *NdArray::random(bool cuda, Shape shape, float mean, float stddev)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->random(mean, stddev);

    return arr;
}

NdArray *NdArray::random_ints(bool cuda, Shape shape, int upper_bound)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->random_ints(upper_bound);

    return arr;
}

NdArray *NdArray::one_hot(NdArray *src)
{
    int lst_dim_idx = src->num_dims() - 1;

    if (src->shape()[lst_dim_idx] != 1)
    {
        THROW_ERROR("NDARRAY ONE HOT ERROR: last dimension must be 1");
    }

    float min_val = src->min();

    if (min_val < 0.0f)
    {
        THROW_ERROR("NDARRAY ONE HOT ERROR: negative numbers not allowed");
    }

    int max_val = src->max();
    int oh_dim = ((int)max_val) + 1;

    std::vector<int> dst_dims = src->shape().dims();
    dst_dims[lst_dim_idx] = oh_dim;

    NdArray *dst = NdArray::zeros(src->is_cuda(), Shape(dst_dims));

    for (int i = 0; i < src->count(); i++)
    {
        int val = (int)src->get_val(i);
        dst->set_val(i * oh_dim + val, 1.0f);
    }

    return dst;
}

NdArray *NdArray::pad(NdArray *src, int pad_row_cnt, int pad_col_cnt)
{
    if (src->num_dims() < 2)
    {
        THROW_ERROR("NDARRAY PAD ERROR: shape must have at least 2 dimensions");
    }

    bool orig_cuda = src->cuda_;
    src->to_cuda();

    int col_dim_idx = src->num_dims() - 1;
    int row_dim_idx = col_dim_idx - 1;

    int src_row_cnt = src->shape()[row_dim_idx];
    int src_col_cnt = src->shape()[col_dim_idx];

    std::vector<int> dst_dims;
    for (int i = 0; i < row_dim_idx; i++)
    {
        dst_dims.push_back(src->shape()[i]);
    }

    int dst_row_cnt = src_row_cnt + (pad_row_cnt * 2);
    int dst_col_cnt = src_col_cnt + (pad_col_cnt * 2);

    dst_dims.push_back(dst_row_cnt);
    dst_dims.push_back(dst_col_cnt);

    NdArray *dst = NdArray::zeros(src->cuda_, Shape(dst_dims));

    int grid_row_cnt = (src_row_cnt / CUDA_THREADS_PER_BLOCK) + 1;
    int grid_col_cnt = (src_col_cnt / CUDA_THREADS_PER_BLOCK) + 1;

    dim3 grid_dims(grid_col_cnt, grid_row_cnt);
    dim3 block_dims(CUDA_THREADS_PER_BLOCK, CUDA_THREADS_PER_BLOCK);

    switch (src->num_dims())
    {
    case 2:
    {
        float *src_data = src->data();
        float *dst_data = dst->data();

        k_pad<<<grid_dims, block_dims>>>(dst_data, src_data, dst_row_cnt, dst_col_cnt, src_row_cnt, src_col_cnt,
                                         pad_row_cnt, pad_col_cnt);
    }
    break;
    case 3:
    {
        for (int i = 0; i < src->shape()[0]; i++)
        {
            float *src_data = &src->data()[(i * src_row_cnt * src_col_cnt)];
            float *dst_data = &dst->data()[(i * dst_row_cnt * dst_col_cnt)];

            k_pad<<<grid_dims, block_dims>>>(dst_data, src_data, dst_row_cnt, dst_col_cnt, src_row_cnt, src_col_cnt,
                                             pad_row_cnt, pad_col_cnt);
        }
    }
    break;
    case 4:
    {
        for (int i = 0; i < src->shape()[0]; i++)
        {
            for (int j = 0; j < src->shape()[1]; j++)
            {
                float *src_data = &src->data()[(i * src->shape()[1] * src_row_cnt * src_col_cnt) + (j * src_row_cnt * src_col_cnt)];
                float *dst_data = &dst->data()[(i * dst->shape()[1] * dst_row_cnt * dst_col_cnt) + (j * dst_row_cnt * dst_col_cnt)];

                k_pad<<<grid_dims, block_dims>>>(dst_data, src_data, dst_row_cnt, dst_col_cnt, src_row_cnt, src_col_cnt,
                                                 pad_row_cnt, pad_col_cnt);
            }
        }
    }
    break;
    default:
        THROW_ERROR("NDARRAY PAD ERROR: shape must not have more than 4 dimensions");
        break;
    }

    if (!orig_cuda)
    {
        src->to_cpu();
        dst->to_cpu();
    }

    return dst;
}

void NdArray::print_vec(float *data, int cnt)
{
    printf("[ ");
    for (int i = 0; i < cnt; i++)
    {
        float val = data[i];

        if (i == cnt - 1)
        {
            if (val >= 0.0f)
            {
                printf(" %f ", val);
            }
            else
            {
                printf("%f ", val);
            }
        }
        else
        {
            if (val >= 0.0f)
            {
                printf(" %f\t", val);
            }
            else
            {
                printf("%f\t", val);
            }
        }
    }
    printf(" ]");
}

void NdArray::print_mtx(float *data, int row_cnt, int col_cnt, const char *whitespace_str)
{
    printf("%s[\n", whitespace_str);
    for (int i = 0; i < row_cnt; i++)
    {
        printf("%s   ", whitespace_str);

        NdArray::print_vec(&data[i * col_cnt], col_cnt);

        printf("\n");
    }
    printf("%s]\n", whitespace_str);
}

void NdArray::print()
{
    bool orig_cuda = this->cuda_;
    this->to_cpu();

    this->shape_.print();
    printf("\n");

    switch (this->num_dims())
    {
    case 1:
    {
        NdArray::print_vec(this->data_, this->count());
    }
    break;
    case 2:
    {
        NdArray::print_mtx(this->data_, this->shape_[0], this->shape_[1], "");
    }
    break;
    case 3:
    {
        int mtx_cnt = this->shape_[0];
        int row_cnt = this->shape_[1];
        int col_cnt = this->shape_[2];

        printf("[\n");
        for (int i = 0; i < mtx_cnt; i++)
        {
            NdArray::print_mtx(&this->data_[i * row_cnt * col_cnt], row_cnt, col_cnt, "   ");
        }
        printf("]");
    }
    break;
    case 4:
    {
        int mtx_cnt = this->shape_[1];
        int row_cnt = this->shape_[2];
        int col_cnt = this->shape_[3];

        printf("[\n");
        for (int i = 0; i < this->shape_[0]; i++)
        {
            printf("   [\n");
            for (int j = 0; j < mtx_cnt; j++)
            {
                int row_cnt = this->shape_[2];
                int col_cnt = this->shape_[3];

                NdArray::print_mtx(&this->data_[(i * mtx_cnt * row_cnt * col_cnt) + (j * row_cnt * col_cnt)],
                                   row_cnt, col_cnt, "      ");
            }
            printf("   ]\n");
        }
        printf("]");
    }
    break;
    default:
        break;
    }

    printf("\n");

    if (orig_cuda)
    {
        this->to_cuda();
    }
}

void NdArray::copy(NdArray *src)
{
    this->cuda_ = src->cuda_;
    this->shape_ = src->shape_;
    hipMemcpy(this->data_, src->data_, src->size(), hipMemcpyDefault);
}

void NdArray::reshape(Shape shape)
{
    this->shape_ = shape;

    if (this->cuda_)
    {
        hipFree(this->data_);
        hipMalloc(&this->data_, this->size());
    }
    else
    {
        free(this->data_);
        this->data_ = (float *)malloc(this->size());
    }
}

void NdArray::change_dim(int dim_idx, int dim)
{
    std::vector<int> dims = this->shape_.dims();
    dims[dim_idx] = dim;
    this->reshape(Shape(dims));
}

bool NdArray::is_cuda()
{
    return this->cuda_;
}

void NdArray::to_cpu()
{
    if (this->cuda_)
    {
        size_t size = this->size();
        float *dst = (float *)malloc(size);
        hipMemcpy(dst, this->data_, size, hipMemcpyDeviceToHost);
        hipFree(this->data_);
        this->data_ = dst;
        this->cuda_ = false;
    }
}

void NdArray::to_cuda()
{
    if (!this->cuda_)
    {
        size_t size = this->size();
        float *dst;
        hipMalloc(&dst, size);
        hipMemcpy(dst, this->data_, size, hipMemcpyHostToDevice);
        free(this->data_);
        this->data_ = dst;
        this->cuda_ = true;
    }
}

Shape NdArray::shape()
{
    return this->shape_;
}

int NdArray::num_dims()
{
    return this->shape_.num_dims();
}

int NdArray::dims_size()
{
    return this->shape_.dims_size();
}

size_t NdArray::size()
{
    return sizeof(float) * this->dims_size();
}

int NdArray::count()
{
    return this->dims_size();
}

float NdArray::sum()
{
    float sum_val = 0.0f;

    for (int i = 0; i < this->count(); i++)
    {
        sum_val += this->get_val(i);
    }

    return sum_val;
}

float NdArray::min()
{
    float min_val = FLT_MAX;

    float val = 0;

    for (int i = 0; i < this->count(); i++)
    {
        val = this->get_val(i);

        if (val < min_val)
        {
            min_val = val;
        }
    }

    return min_val;
}

float NdArray::max()
{
    float max_val = -FLT_MAX;

    float val = 0;

    for (int i = 0; i < this->count(); i++)
    {
        val = this->get_val(i);

        if (val > max_val)
        {
            max_val = val;
        }
    }

    return max_val;
}

float NdArray::mean()
{
    return this->sum() / this->count();
}

float NdArray::stddev()
{
    float stddev_val = 0.0f;

    float mean_val = this->mean();

    for (int i = 0; i < this->count(); i++)
    {
        float diff = this->get_val(i) - mean_val;
        stddev_val = diff * diff;
    }

    stddev_val /= this->count();

    return sqrt(stddev_val);
}

float NdArray::get_val(int idx)
{
    float val;
    hipMemcpy(&val, &this->data_[idx], sizeof(float), hipMemcpyDefault);
    return val;
}

void NdArray::set_val(int idx, float val)
{
    hipMemcpy(&this->data_[idx], &val, sizeof(float), hipMemcpyDefault);
}

float *NdArray::data()
{
    return this->data_;
}

void NdArray::zeros()
{
    size_t size = this->size();

    if (this->cuda_)
    {
        hipMemset(this->data_, 0, size);
    }
    else
    {
        memset(this->data_, 0, size);
    }
}

void NdArray::ones()
{
    if (this->is_cuda())
    {
        k_set_all<<<(this->count() / CUDA_THREADS_PER_BLOCK + 1), CUDA_THREADS_PER_BLOCK>>>(this->data_, this->count(), 1.0f);
    }
    else
    {
        for (int i = 0; i < this->count(); i++)
        {
            this->data_[i] = 1.0f;
        }
    }
}

void NdArray::full(float val)
{
    if (this->cuda_)
    {
        k_set_all<<<this->count() / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(this->data_, this->count(), val);
    }
    else
    {
        for (int i = 0; i < this->count(); i++)
        {
            this->data_[i] = val;
        }
    }
}

void NdArray::random(float mean, float stddev)
{
    std::random_device rd;
    std::mt19937 gen(rd());

    for (int i = 0; i < this->count(); i++)
    {
        std::normal_distribution<float> d(mean, stddev);
        this->set_val(i, d(gen));
    }
}

void NdArray::random_ints(int upper_bound)
{
    for (int i = 0; i < this->count(); i++)
    {
        this->set_val(i, rand() % upper_bound);
    }
}
