#include "hip/hip_runtime.h"
#include "util.cuh"

using namespace zero::core;

CpuStopWatch::CpuStopWatch()
{
    this->beg_ = 0;
    this->end_ = 0;
}

CpuStopWatch::~CpuStopWatch()
{
}

void CpuStopWatch::start()
{
    this->beg_ = clock();
    this->end_ = this->beg_;
}

void CpuStopWatch::stop()
{
    this->end_ = clock();
}

double CpuStopWatch::get_elapsed_seconds()
{
    return ((double)(this->end_ - this->beg_)) / CLOCKS_PER_SEC;
}

void CpuStopWatch::print_elapsed_seconds()
{
    printf("ELAPSED SECONDS: %f\n", this->get_elapsed_seconds());
}

CudaStopWatch::CudaStopWatch()
{
    hipEventCreate(&this->beg_);
    hipEventCreate(&this->end_);
}

CudaStopWatch::~CudaStopWatch()
{
}

void CudaStopWatch::start()
{
    hipEventRecord(this->beg_, 0);
}

void CudaStopWatch::stop()
{
    hipDeviceSynchronize();
    hipEventRecord(this->end_, 0);
    hipEventSynchronize(this->end_);
}

double CudaStopWatch::get_elapsed_seconds()
{
    float elapsed_ms;
    hipEventElapsedTime(&elapsed_ms, this->beg_, this->end_);

    return ((double)elapsed_ms / 1000.0);
}

void CudaStopWatch::print_elapsed_seconds()
{
    printf("ELAPSED SECONDS: %f\n", this->get_elapsed_seconds());
}

long long FileUtils::get_file_size(const char *name)
{

    HANDLE hFile = CreateFile((LPCSTR)name, GENERIC_READ,
                              FILE_SHARE_READ | FILE_SHARE_WRITE, NULL, OPEN_EXISTING,
                              FILE_ATTRIBUTE_NORMAL, NULL);
    if (hFile == INVALID_HANDLE_VALUE)
        return -1; // error condition, could call GetLastError to find out more

    LARGE_INTEGER size;
    if (!GetFileSizeEx(hFile, &size))
    {
        CloseHandle(hFile);
        return -1; // error condition, could call GetLastError to find out more
    }

    CloseHandle(hFile);
    return size.QuadPart;
}