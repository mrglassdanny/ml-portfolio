#include "hip/hip_runtime.h"
#include "hadamard_product.cuh"

using namespace zero::core;
using namespace zero::nn::layer;

__global__ void k_hadamard_product_evaluate(float *in, float *w, float *out, int batch_size, int channel_cnt, int row_cnt, int col_cnt,
                                            int filter_cnt)
{
    int f_r_c_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_idx = blockIdx.y * blockDim.y + threadIdx.y;

    int filter_idx = f_r_c_idx / (row_cnt * col_cnt);
    int row_idx = (f_r_c_idx - (filter_idx * (row_cnt * col_cnt))) / col_cnt;
    int col_idx = f_r_c_idx % col_cnt;

    if (filter_idx < filter_cnt && row_idx < row_cnt && col_idx < col_cnt && batch_idx < batch_size)
    {
        int in_cnt = row_cnt * col_cnt;
        int w_cnt = in_cnt;
        int out_cnt = in_cnt;

        float *l_in = &in[(batch_idx * channel_cnt * in_cnt)];
        float *l_w = &w[(filter_idx * channel_cnt * w_cnt)];
        float *l_out = &out[((batch_idx * filter_cnt * out_cnt) + (filter_idx * out_cnt))];

        for (int channel_idx = 0; channel_idx < channel_cnt; channel_idx++)
        {
            l_out[row_idx * col_cnt + col_idx] +=
                (l_in[(channel_idx * in_cnt) + (row_idx * col_cnt) + col_idx] *
                 l_w[(channel_idx * w_cnt) + (row_idx * col_cnt) + col_idx]);
        }
    }
}

__global__ void k_hadamard_product_inc_param_derivatives(float *in, float *in_n, float *n, float *dw, int batch_size, int channel_cnt, int filter_cnt,
                                                         int row_cnt, int col_cnt, int cnt)
{
    int c_r_c_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int filter_idx = blockIdx.y * blockDim.y + threadIdx.y;

    int channel_idx = c_r_c_idx / cnt;
    int row_idx = (c_r_c_idx - (channel_idx * cnt)) / col_cnt;
    int col_idx = c_r_c_idx % col_cnt;

    if (channel_idx < channel_cnt && row_idx < row_cnt && col_idx < col_cnt && filter_idx < filter_cnt)
    {
        int w_elem_idx = (filter_idx * channel_cnt * cnt) + (channel_idx * cnt) + (row_idx * col_cnt + col_idx);
        int b_elem_idx = filter_idx * channel_cnt + channel_idx;

        for (int batch_idx = 0; batch_idx < batch_size; batch_idx++)
        {
            int in_elem_idx = (batch_idx * filter_cnt * cnt) + (filter_idx * cnt) + (row_idx * col_cnt + col_idx);

            dw[w_elem_idx] +=
                (in[in_elem_idx] * n[(batch_idx * channel_cnt * cnt) + (channel_idx * cnt) + ((row_idx * col_cnt) + col_idx)]);
        }
    }
}

__global__ void k_hadamard_product_agg_derivatives(float *in, float *w, float *out, int batch_size, int channel_cnt, int filter_cnt,
                                                   int row_cnt, int col_cnt, int cnt)
{
    int f_r_c_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_idx = blockIdx.y * blockDim.y + threadIdx.y;

    int filter_idx = f_r_c_idx / cnt;
    int row_idx = (f_r_c_idx - (filter_idx * cnt)) / col_cnt;
    int col_idx = f_r_c_idx % col_cnt;

    if (filter_idx < filter_cnt && row_idx < row_cnt && col_idx < col_cnt && batch_idx < batch_size)
    {
        int in_elem_idx = (batch_idx * filter_cnt * cnt) + (filter_idx * cnt) + (row_idx * col_cnt + col_idx);

        for (int channel_idx = 0; channel_idx < channel_cnt; channel_idx++)
        {
            int out_elem_idx = (batch_idx * channel_cnt * cnt) + (channel_idx * cnt) + (row_idx * col_cnt) + col_idx;

            atomicAdd(&out[out_elem_idx], (in[in_elem_idx] * w[(filter_idx * channel_cnt * cnt) + (channel_idx * cnt) + (row_idx * col_cnt + col_idx)]));
        }
    }
}

HadamardProduct::HadamardProduct(Shape in_shape, int filter_cnt, ActivationType activation)
{
    this->n_ = new Tensor(true, in_shape);
    this->dn_ = new Tensor(true, in_shape);
    this->params_ = new Parameters(Shape(filter_cnt, this->channels(), this->rows(), this->cols()),
                                   Shape(filter_cnt, this->channels()), this->rows(), this->cols());

    this->activation_ = activation;
}

void HadamardProduct::evaluate(Tensor *out)
{
    int grid_row_cnt = (this->batch_size() / ZERO_CORE_CUDA_THREADS_PER_BLOCK) + 1;
    int grid_col_cnt = ((this->filters() * this->rows() * this->cols()) / ZERO_CORE_CUDA_THREADS_PER_BLOCK) + 1;

    dim3 grid_dims(grid_col_cnt, grid_row_cnt);
    dim3 block_dims(ZERO_CORE_CUDA_THREADS_PER_BLOCK, ZERO_CORE_CUDA_THREADS_PER_BLOCK);

    Tensor *n = this->n_;
    Tensor *w = this->params_->weights();

    k_hadamard_product_evaluate<<<grid_dims, block_dims>>>(n->data(), w->data(), out->data(), this->batch_size(), this->channels(), this->rows(), this->cols(),
                                                           this->filters());

    Activation::evaluate(out, this->batch_size(), this->out_features(), this->activation_);
}

void HadamardProduct::derive(Tensor *in, Tensor *in_n)
{
    Tensor *n = this->n_;
    Tensor *dn = this->dn_;
    Tensor *w = this->params_->weights();
    Tensor *b = this->params_->biases();
    Tensor *dw = this->params_->weight_gradients();

    Activation::derive(in, in_n, this->batch_size(), this->out_features(), this->activation_);

    {
        int grid_row_cnt = (this->filters() / ZERO_CORE_CUDA_THREADS_PER_BLOCK) + 1;
        int grid_col_cnt = ((this->channels() * this->rows() * this->cols()) / ZERO_CORE_CUDA_THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(ZERO_CORE_CUDA_THREADS_PER_BLOCK, ZERO_CORE_CUDA_THREADS_PER_BLOCK);

        k_hadamard_product_inc_param_derivatives<<<grid_dims, block_dims>>>(in->data(), in_n->data(), n->data(), dw->data(), this->batch_size(), this->channels(), this->filters(),
                                                                            this->rows(), this->cols(), (this->rows() * this->cols()));
    }

    {
        int grid_row_cnt = (this->batch_size() / ZERO_CORE_CUDA_THREADS_PER_BLOCK) + 1;
        int grid_col_cnt = ((this->filters() * this->rows() * this->cols()) / ZERO_CORE_CUDA_THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(ZERO_CORE_CUDA_THREADS_PER_BLOCK, ZERO_CORE_CUDA_THREADS_PER_BLOCK);

        k_hadamard_product_agg_derivatives<<<grid_dims, block_dims>>>(in->data(), w->data(), dn->data(), this->batch_size(), this->channels(), this->filters(),
                                                                      this->rows(), this->cols(), (this->rows() * this->cols()));
    }
}

Shape HadamardProduct::input_shape()
{
    return this->n_->shape();
}

Shape HadamardProduct::output_shape()
{
    return Shape(this->batch_size(), this->filters(), this->rows(), this->cols());
}

void HadamardProduct::validate()
{
    if (this->input_shape().num_dims() != 4)
    {
        ZERO_CORE_THROW_ERROR("HADAMARD_PRODUCT LAYER VALIDATION FAILED: invalid input shape");
    }
}

void HadamardProduct::summarize()
{
    std::string cls_name(typeid(*this).name());
    for (int i = cls_name.size(); i < 26; i++)
    {
        cls_name.push_back(' ');
    }

    printf("%s\t", cls_name.c_str());

    this->input_shape().print_pad(16, true);

    printf(" -> ");
    this->output_shape().print_pad(16, false);

    printf("\tFilters: %d\t", this->filters());

    Activation::summarize(this->activation_);
}

int HadamardProduct::channels()
{
    return this->n_->shape()[1];
}

int HadamardProduct::rows()
{
    return this->n_->shape()[2];
}

int HadamardProduct::cols()
{
    return this->n_->shape()[3];
}

int HadamardProduct::filters()
{
    return this->params_->weights()->shape()[0];
}