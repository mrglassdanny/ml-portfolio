#include "hip/hip_runtime.h"
#include "model.cuh"

using namespace nn;

Model::Model()
{
    this->loss_ = nullptr;
    this->optim_ = nullptr;

    this->validations_ = Validations{false, false, false};
}

Model::~Model()
{
    for (Layer *lyr : this->lyrs_)
    {
        delete lyr;
    }

    if (this->loss_ != nullptr)
    {
        delete this->loss_;
    }

    if (this->optim_ != nullptr)
    {
        delete this->optim_;
    }
}

NdArray *Model::forward(NdArray *x)
{
    this->reset_layer_shapes();

    this->validate_layers();
    this->validate_input(x);

    x->to_cuda();

    this->first_layer()->copy_neurons(x);

    for (int i = 0; i < this->lyrs_.size() - 1; i++)
    {
        Layer *lyr = this->lyrs_[i];
        Layer *nxt_lyr = this->lyrs_[i + 1];

        nxt_lyr->neurons()->zeros();
        lyr->evaluate(nxt_lyr->neurons());
    }

    Layer *lst_lyr = this->last_layer();

    NdArray *p = NdArray::zeros(true, lst_lyr->output_shape());
    lst_lyr->evaluate(p);

    return p;
}

float Model::loss(NdArray *p, NdArray *y)
{
    this->validate_loss();
    this->validate_output(y);

    p->to_cuda();
    y->to_cuda();

    NdArray *losses = NdArray::zeros(true, p->shape());

    this->loss_->evaluate(p, y, losses);

    float mean_loss = losses->sum() / this->batch_size();

    delete losses;
    return mean_loss;
}

void Model::backward(NdArray *p, NdArray *y)
{
    this->validate_layers();
    this->validate_loss();
    this->validate_output(y);

    if (this->loss_ == nullptr)
    {
        return;
    }

    p->to_cuda();
    y->to_cuda();

    NdArray *loss_gradients = this->loss_->derive(p, y);

    for (int i = this->lyrs_.size() - 1; i >= 0; i--)
    {
        loss_gradients = this->lyrs_[i]->derive(loss_gradients);
    }

    delete loss_gradients;
}

void Model::step()
{
    this->validate_optimizer();

    this->optim_->step(this->batch_size());
}

Shape Model::input_shape()
{
    return this->first_layer()->input_shape();
}

Shape Model::output_shape()
{
    return this->last_layer()->output_shape();
}

void Model::validate_layers()
{
    if (this->validations_.layers)
    {
        return;
    }

    if (this->lyrs_.size() == 0)
    {
        THROW_ERROR("MODEL VALIDATION FAILED: no layers");
    }

    for (int i = 0; i < this->lyrs_.size() - 1; i++)
    {
        Layer *lyr = this->lyrs_[i];
        Layer *nxt_lyr = this->lyrs_[i + 1];

        if (lyr->output_shape() != nxt_lyr->input_shape())
        {
            THROW_ERROR("MODEL VALIDATION FAILED: layer output shape does not match next layer input shape");
        }
    }

    for (Layer *lyr : this->lyrs_)
    {
        lyr->validate();
    }

    this->validations_.layers = true;
}

void Model::validate_loss()
{
    if (this->validations_.loss)
    {
        return;
    }

    if (this->loss_ == nullptr)
    {
        THROW_ERROR("MODEL LOSS VALIDATION FAILED: loss not set");
    }

    this->validations_.loss = true;
}

void Model::validate_optimizer()
{
    if (this->validations_.optimizer)
    {
        return;
    }

    if (this->optim_ == nullptr)
    {
        THROW_ERROR("MODEL OPTIMIZER VALIDATION FAILED: optimizer not set");
    }

    this->validations_.optimizer = true;
}

void Model::validate_input(NdArray *x)
{
    if (this->input_shape() != x->shape())
    {
        THROW_ERROR("MODEL INPUT VALIDATION FAILED: X shape does not match model input shape");
    }
}

void Model::validate_output(NdArray *y)
{
    if (this->output_shape() != y->shape())
    {
        THROW_ERROR("MODEL OUTPUT VALIDATION FAILED: Y shape does not match model output shape");
    }
}

void Model::validate_gradients(NdArray *x, NdArray *y, bool print_params)
{
    this->validate_layers();
    this->validate_loss();
    this->validate_input(x);
    this->validate_output(y);

    x->to_cuda();
    y->to_cuda();

    float agg_ana_grad = 0.0f;
    float agg_num_grad = 0.0f;
    float agg_grad_diff = 0.0f;

    NdArray *p = this->forward(x);
    this->backward(p, y);
    delete p;

    int param_idx = 0;
    for (Parameters *params : this->parameters())
    {
        NdArray *w = params->weights();
        NdArray *b = params->biases();
        NdArray *dw = params->weight_gradients();
        NdArray *db = params->bias_gradients();

        for (int i = 0; i < w->count(); i++)
        {
            float w_val = w->get_val(i);

            w->set_val(i, w_val - EPSILON);
            p = this->forward(x);
            float left_loss = this->loss(p, y);
            delete p;

            w->set_val(i, w_val + EPSILON);
            p = this->forward(x);
            float right_loss = this->loss(p, y);
            delete p;

            w->set_val(i, w_val);

            float num_grad = (right_loss - left_loss) / (2.0f * EPSILON);
            float ana_grad = dw->get_val(i);

            if (print_params)
            {
                printf("W: %d  %d\t|%f - %f| = %f\n", param_idx, i, ana_grad, num_grad, fabs(ana_grad - num_grad));
            }

            agg_ana_grad += (ana_grad * ana_grad);
            agg_num_grad += (num_grad * num_grad);
            agg_grad_diff += ((ana_grad - num_grad) * (ana_grad - num_grad));
        }

        for (int i = 0; i < b->count(); i++)
        {
            float b_val = b->get_val(i);

            b->set_val(i, b_val - EPSILON);
            p = this->forward(x);
            float left_loss = this->loss(p, y);
            delete p;

            b->set_val(i, b_val + EPSILON);
            p = this->forward(x);
            float right_loss = this->loss(p, y);
            delete p;

            b->set_val(i, b_val);

            float num_grad = (right_loss - left_loss) / (2.0f * EPSILON);
            float ana_grad = db->get_val(i);

            if (print_params)
            {
                printf("B: %d  %d\t|%f - %f| = %f\n", param_idx, i, ana_grad, num_grad, fabs(ana_grad - num_grad));
            }

            agg_ana_grad += (ana_grad * ana_grad);
            agg_num_grad += (num_grad * num_grad);
            agg_grad_diff += ((ana_grad - num_grad) * (ana_grad - num_grad));
        }

        param_idx++;
    }

    if ((agg_grad_diff) == 0.0f && (agg_ana_grad + agg_num_grad) == 0.0f)
    {
        printf("GRADIENT CHECK RESULT: %f\n", 0.0f);
    }
    else
    {
        printf("GRADIENT CHECK RESULT: %f\n", (agg_grad_diff) / (agg_ana_grad + agg_num_grad));

        if ((agg_grad_diff) / (agg_ana_grad + agg_num_grad) > EPSILON)
        {
            THROW_ERROR("MODEL GRADIENTS VALIDATION FAILED");
        }
    }
}

void Model::summarize()
{
    printf("=========================== MODEL SUMMARY ===========================\n");

    printf("\nLayers: (%d)\n", this->lyrs_.size());
    for (int i = 0; i < this->lyrs_.size(); i++)
    {
        printf("\t%d\t", i + 1);
        this->lyrs_[i]->summarize();
        printf("\n");
    }
    printf("\n");

    printf("Loss: ");
    if (this->loss_ != nullptr)
    {
        this->loss_->summarize();
    }
    else
    {
        printf("None");
    }
    printf("\n\n");

    printf("Optimizer: ");
    if (this->optim_ != nullptr)
    {
        this->optim_->summarize();
    }
    else
    {
        printf("None");
    }
    printf("\n\n");

    printf("=====================================================================\n");
}

void Model::add_layer(Layer *lyr)
{
    this->lyrs_.push_back(lyr);
}

void Model::set_loss(Loss *loss)
{
    this->loss_ = loss;
}

void Model::set_optimizer(Optimizer *optim)
{
    this->optim_ = optim;
}

void Model::linear(int out_feature_cnt)
{
    this->add_layer(new Linear(this->output_shape(), Shape(this->batch_size(), out_feature_cnt)));
}

void Model::linear(Shape y_shape)
{
    this->add_layer(new Linear(this->output_shape(), y_shape));
}

void Model::linear(int batch_size, int in_feature_cnt, int out_feature_cnt)
{
    this->add_layer(new Linear(Shape(batch_size, in_feature_cnt), Shape(batch_size, out_feature_cnt)));
}

void Model::conv2d(Shape filter_shape)
{
    this->add_layer(new Conv2d(this->output_shape(), filter_shape, Padding{0, 0}, Stride{1, 1}));
}

void Model::conv2d(Shape filter_shape, Stride stride)
{
    this->add_layer(new Conv2d(this->output_shape(), filter_shape, Padding{0, 0}, stride));
}

void Model::conv2d(Shape filter_shape, Padding padding, Stride stride)
{
    this->add_layer(new Conv2d(this->output_shape(), filter_shape, padding, stride));
}

void Model::conv2d(Shape in_shape, Shape filter_shape, Stride stride)
{
    this->add_layer(new Conv2d(in_shape, filter_shape, Padding{0, 0}, stride));
}

void Model::conv2d(Shape in_shape, Shape filter_shape, Padding padding, Stride stride)
{
    this->add_layer(new Conv2d(in_shape, filter_shape, padding, stride));
}

void Model::sigmoid()
{
    this->add_layer(new Sigmoid(this->output_shape()));
}

void Model::tanh()
{
    this->add_layer(new Tanh(this->output_shape()));
}

void Model::relu()
{
    this->add_layer(new ReLU(this->output_shape()));
}

std::vector<Layer *> Model::layers()
{
    return this->lyrs_;
}

std::vector<Parameters *> Model::parameters()
{
    std::vector<Parameters *> params;

    for (Layer *lyr : this->lyrs_)
    {
        if (Learnable *lrn = dynamic_cast<Learnable *>(lyr))
        {
            params.push_back(lrn->parameters());
        }
    }

    return params;
}

Layer *Model::first_layer()
{
    return this->lyrs_[0];
}

Layer *Model::last_layer()
{
    return this->lyrs_[this->lyrs_.size() - 1];
}

void Model::reset_layer_shapes()
{
    for (Layer *lyr : this->lyrs_)
    {
        lyr->reset_shape();
    }
}

int Model::batch_size()
{
    return this->first_layer()->batch_size();
}

void Model::performance_check(NdArray *x, NdArray *y, int epoch_cnt)
{
    CudaStopWatch *sw = new CudaStopWatch();

    sw->start();

    for (int i = 0; i < epoch_cnt; i++)
    {
        NdArray *p = this->forward(x);
        this->loss(p, y);
        this->backward(p, y);
        this->step();
        delete p;
    }

    sw->stop();

    sw->print_elapsed_seconds();

    delete sw;
}
