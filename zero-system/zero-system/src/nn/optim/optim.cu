#include "hip/hip_runtime.h"
#include "optim.cuh"

using namespace nn::optim;
using namespace nn::layer;

__global__ void k_sgd_weight_step(float *w, float *dw, int w_cnt, float lr, int batch_size)
{
    int w_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (w_elem_idx < w_cnt)
    {
        w[w_elem_idx] -= (lr * dw[w_elem_idx] / batch_size);
    }
}

__global__ void k_sgd_bias_step(float *b, float *db, int b_cnt, float lr, int batch_size)
{
    int b_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (b_elem_idx < b_cnt)
    {
        b[b_elem_idx] -= (lr * db[b_elem_idx] / batch_size);
    }
}

__global__ void k_sgd_momentum_weight_step(float *w, float *dw, float *mdw, int w_cnt, float lr, float beta1, int step_num, int batch_size)
{
    int w_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (w_elem_idx < w_cnt)
    {
        mdw[w_elem_idx] = beta1 * mdw[w_elem_idx] + (1.0f - beta1) * dw[w_elem_idx];

        float corrected_mdw = mdw[w_elem_idx] / (1.0f - pow(beta1, step_num));

        w[w_elem_idx] -= (lr * corrected_mdw / batch_size);
    }
}

__global__ void k_sgd_momentum_bias_step(float *b, float *db, float *mdb, int b_cnt, float lr, float beta1, int step_num, int batch_size)
{
    int b_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (b_elem_idx < b_cnt)
    {
        mdb[b_elem_idx] = beta1 * mdb[b_elem_idx] + (1.0f - beta1) * db[b_elem_idx];

        float corrected_mdb = mdb[b_elem_idx] / (1.0f - pow(beta1, step_num));

        b[b_elem_idx] -= (lr * corrected_mdb / batch_size);
    }
}

__global__ void k_adam_weight_step(float *w, float *dw, float *mdw, float *vdw, int w_cnt, float lr, float beta1, float beta2, int step_num, int batch_size)
{
    int w_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (w_elem_idx < w_cnt)
    {
        mdw[w_elem_idx] = beta1 * mdw[w_elem_idx] + (1.0f - beta1) * dw[w_elem_idx];
        vdw[w_elem_idx] = beta2 * vdw[w_elem_idx] + (1.0f - beta2) * (dw[w_elem_idx] * dw[w_elem_idx]);

        float corrected_mdw = mdw[w_elem_idx] / (1.0f - pow(beta1, step_num));
        float corrected_vdw = vdw[w_elem_idx] / (1.0f - pow(beta2, step_num));

        w[w_elem_idx] -= (lr * (corrected_mdw / (sqrt(corrected_vdw) + EPSILON)) / batch_size);
    }
}

__global__ void k_adam_bias_step(float *b, float *db, float *mdb, float *vdb, int b_cnt, float lr, float beta1, float beta2, int step_num, int batch_size)
{
    int b_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (b_elem_idx < b_cnt)
    {
        mdb[b_elem_idx] = beta1 * mdb[b_elem_idx] + (1.0f - beta1) * db[b_elem_idx];
        vdb[b_elem_idx] = beta2 * vdb[b_elem_idx] + (1.0f - beta2) * (db[b_elem_idx] * db[b_elem_idx]);

        float corrected_mdb = mdb[b_elem_idx] / (1.0f - pow(beta1, step_num));
        float corrected_vdb = vdb[b_elem_idx] / (1.0f - pow(beta2, step_num));

        b[b_elem_idx] -= (lr * (corrected_mdb / (sqrt(corrected_vdb) + EPSILON)) / batch_size);
    }
}

Optimizer::Optimizer(std::vector<Parameters *> model_params, float learning_rate)
{
    this->model_params_ = model_params;
    this->lr_ = learning_rate;
}

void Optimizer::summarize()
{
    std::string cls_name(typeid(*this).name());
    printf("%s", cls_name.c_str());

    size_t params_cnt = 0;
    for (Parameters *params : this->model_params_)
    {
        params_cnt += params->count();
    }

    printf("\n\tParameters: %zd\t\tLearning rate: %f", params_cnt, this->lr_);
}

SGD::SGD(std::vector<Parameters *> model_params, float learning_rate)
    : Optimizer(model_params, learning_rate)
{
}

void SGD::step(int batch_size)
{
    for (Parameters *params : this->model_params_)
    {
        NdArray *w = params->weights();
        NdArray *b = params->biases();
        NdArray *dw = params->weight_gradients();
        NdArray *db = params->bias_gradients();

        int w_cnt = w->count();
        int b_cnt = b->count();

        k_sgd_weight_step<<<w_cnt / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(w->data(), dw->data(), w_cnt, this->lr_, batch_size);
        k_sgd_bias_step<<<b_cnt / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(b->data(), db->data(), b_cnt, this->lr_, batch_size);

        params->zero_grad();
        this->step_num_++;
    }
}

SGDMomentum::SGDMomentum(std::vector<Parameters *> model_params, float learning_rate, float beta1)
    : Optimizer(model_params, learning_rate)
{
    this->beta1_ = beta1;

    for (Parameters *params : model_params)
    {
        this->mdws_.push_back(NdArray::zeros(true, params->weight_gradients()->shape()));
        this->mdbs_.push_back(NdArray::zeros(true, params->bias_gradients()->shape()));
    }
}

SGDMomentum::~SGDMomentum()
{
    for (int i = 0; i < this->mdws_.size(); i++)
    {
        delete this->mdws_[i];
        delete this->mdbs_[i];
    }
}

void SGDMomentum::step(int batch_size)
{
    for (int i = 0; i < this->model_params_.size(); i++)
    {
        Parameters *params = this->model_params_[i];

        NdArray *w = params->weights();
        NdArray *b = params->biases();
        NdArray *dw = params->weight_gradients();
        NdArray *db = params->bias_gradients();
        NdArray *mdw = this->mdws_[i];
        NdArray *mdb = this->mdbs_[i];

        int w_cnt = w->count();
        int b_cnt = b->count();

        k_sgd_momentum_weight_step<<<w_cnt / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(w->data(), dw->data(), mdw->data(),
                                                                                                   w_cnt, this->lr_, this->beta1_, this->step_num_, batch_size);
        k_sgd_momentum_bias_step<<<b_cnt / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(b->data(), db->data(), mdb->data(),
                                                                                                 b_cnt, this->lr_, this->beta1_, this->step_num_, batch_size);

        params->zero_grad();
        this->step_num_++;
    }
}

Adam::Adam(std::vector<Parameters *> model_params, float learning_rate, float beta1, float beta2)
    : Optimizer(model_params, learning_rate)
{
    this->beta1_ = beta1;
    this->beta2_ = beta2;

    for (Parameters *params : model_params)
    {
        this->mdws_.push_back(NdArray::zeros(true, params->weight_gradients()->shape()));
        this->mdbs_.push_back(NdArray::zeros(true, params->bias_gradients()->shape()));
        this->vdws_.push_back(NdArray::zeros(true, params->weight_gradients()->shape()));
        this->vdbs_.push_back(NdArray::zeros(true, params->bias_gradients()->shape()));
    }
}

Adam::~Adam()
{
    for (int i = 0; i < this->mdws_.size(); i++)
    {
        delete this->mdws_[i];
        delete this->mdbs_[i];
        delete this->vdws_[i];
        delete this->vdbs_[i];
    }
}

void Adam::step(int batch_size)
{
    for (int i = 0; i < this->model_params_.size(); i++)
    {
        Parameters *params = this->model_params_[i];

        NdArray *w = params->weights();
        NdArray *b = params->biases();
        NdArray *dw = params->weight_gradients();
        NdArray *db = params->bias_gradients();
        NdArray *vdw = this->mdws_[i];
        NdArray *vdb = this->mdbs_[i];
        NdArray *sdw = this->vdws_[i];
        NdArray *sdb = this->vdbs_[i];

        int w_cnt = w->count();
        int b_cnt = b->count();

        k_adam_weight_step<<<w_cnt / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(w->data(), dw->data(), vdw->data(), sdw->data(),
                                                                                           w_cnt, this->lr_, this->beta1_, this->beta2_, this->step_num_, batch_size);
        k_adam_bias_step<<<b_cnt / CUDA_THREADS_PER_BLOCK + 1, CUDA_THREADS_PER_BLOCK>>>(b->data(), db->data(), vdb->data(), sdb->data(),
                                                                                         b_cnt, this->lr_, this->beta1_, this->beta2_, this->step_num_, batch_size);

        params->zero_grad();
        this->step_num_++;
    }
}