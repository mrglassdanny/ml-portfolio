#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>

#include <zero/mod.cuh>

#include "chess.cuh"

using namespace zero::core;
using namespace zero::nn;
using namespace zero::cluster;
using namespace chess;

__global__ void k_co_weight_step(float *w, float *dw, float *mdw, int w_cnt, float lr, float beta1, int step_num, int batch_size)
{
    int w_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (w_elem_idx < w_cnt)
    {
        mdw[w_elem_idx] = beta1 * mdw[w_elem_idx] + (1.0f - beta1) * dw[w_elem_idx];

        float corrected_mdw = mdw[w_elem_idx] / (1.0f - pow(beta1, step_num));

        w[w_elem_idx] -= (lr * corrected_mdw / batch_size);
        if (w[w_elem_idx] < 0.0f)
        {
            w[w_elem_idx] = 0.0f;
        }
    }
}

__global__ void k_co_bias_step(float *b, float *db, float *mdb, int b_cnt, float lr, float beta1, int step_num, int batch_size)
{
    int b_elem_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (b_elem_idx < b_cnt)
    {
        mdb[b_elem_idx] = beta1 * mdb[b_elem_idx] + (1.0f - beta1) * db[b_elem_idx];

        float corrected_mdb = mdb[b_elem_idx] / (1.0f - pow(beta1, step_num));

        b[b_elem_idx] -= (lr * corrected_mdb / batch_size);
        if (b[b_elem_idx] < 0.0f)
        {
            b[b_elem_idx] = 0.0f;
        }
    }
}

class ChessOptimizer : public Optimizer
{
private:
    float beta1_;
    std::vector<Tensor *> mdws_;
    std::vector<Tensor *> mdbs_;

public:
    ChessOptimizer(std::vector<Parameters *> model_params, float learning_rate, float beta1)
        : Optimizer(model_params, learning_rate)
    {
        this->beta1_ = beta1;

        for (Parameters *params : model_params)
        {
            this->mdws_.push_back(Tensor::zeros(true, params->weight_gradients()->shape()));
            this->mdbs_.push_back(Tensor::zeros(true, params->bias_gradients()->shape()));
        }
    }

    ~ChessOptimizer()
    {
        for (int i = 0; i < this->mdws_.size(); i++)
        {
            delete this->mdws_[i];
            delete this->mdbs_[i];
        }
    }

    void step(int batch_size)
    {
        for (int i = 0; i < this->model_params_.size(); i++)
        {
            Parameters *params = this->model_params_[i];

            Tensor *w = params->weights();
            Tensor *b = params->biases();
            Tensor *dw = params->weight_gradients();
            Tensor *db = params->bias_gradients();
            Tensor *mdw = this->mdws_[i];
            Tensor *mdb = this->mdbs_[i];

            int w_cnt = w->count();
            int b_cnt = b->count();

            k_co_weight_step<<<w_cnt / ZERO_CORE_CUDA_THREADS_PER_BLOCK + 1, ZERO_CORE_CUDA_THREADS_PER_BLOCK>>>(w->data(), dw->data(), mdw->data(),
                                                                                                                 w_cnt, this->lr_, this->beta1_, this->step_num_, batch_size);
            k_co_bias_step<<<b_cnt / ZERO_CORE_CUDA_THREADS_PER_BLOCK + 1, ZERO_CORE_CUDA_THREADS_PER_BLOCK>>>(b->data(), db->data(), mdb->data(),
                                                                                                               b_cnt, this->lr_, this->beta1_, this->step_num_, batch_size);
        }

        this->step_num_++;
    }

    Optimizer *ChessOptimizer::copy()
    {
        return new ChessOptimizer(this->model_params_, this->lr_, this->beta1_);
    }
};

class ChessInitializer : public Initializer
{
public:
    void initialize(Tensor *tensor, int fan_in, int fan_out)
    {
        tensor->random(0.0f, sqrt(1.0f / fan_in));
        tensor->abs();
    }

    Initializer *copy()
    {
        return new ChessInitializer();
    }
};

int chess_tanh_accuracy_fn(Tensor *p, Tensor *y, int batch_size)
{
    int correct_cnt = 0;

    for (int i = 0; i < batch_size; i++)
    {
        float p_val = p->get_val(i);
        if (p_val < -0.25f)
        {
            p_val = -1.0f;
        }
        else if (p_val > 0.25f)
        {
            p_val = 1.0f;
        }
        else
        {
            p_val = 0.0f;
        }

        if (p_val == y->get_val(i))
        {
            correct_cnt++;
        }
    }

    return correct_cnt;
}

struct Game
{
    std::vector<Board> boards;
    float lbl;
};

Game self_play(int white_depth, int black_depth, bool print)
{
    Board board;
    Move prev_move;

    Game game;
    game.lbl = 0.0f;

    int move_cnt = 0;

    while (move_cnt < 200)
    {
        if (print)
        {
            printf("\nWHITE TURN\n");
            if (move_cnt == 0)
            {
                board.print();
            }
            else
            {
                board.print(prev_move);
            }
        }

        if (board.is_checkmate(false, true))
        {
            if (print)
                printf("WHITE CHECKMATED!\n");

            game.lbl = -1.0f;
            break;
        }
        else if (!board.has_moves(true))
        {
            if (print)
                printf("WHITE STALEMATED!\n");

            break;
        }

        auto evals = board.minimax_alphabeta_dyn(true, white_depth);
        printf("Ties: %d\n", evals.size());
        int r = rand() % evals.size();
        board.change(evals[r].move);
        prev_move = evals[r].move;
        Board cpy_board;
        cpy_board.copy(&board);
        game.boards.push_back(cpy_board);

        move_cnt++;

        if (print)
        {
            printf("\nBLACK TURN\n");
            board.print(prev_move);
        }

        if (board.is_checkmate(true, true))
        {
            if (print)
                printf("BLACK CHECKMATED!\n");

            game.lbl = 1.0f;
            break;
        }
        else if (!board.has_moves(false))
        {
            if (print)
                printf("BLACK STALEMATED!\n");

            break;
        }

        evals = board.minimax_alphabeta(false, black_depth);
        printf("Ties: %d\n", evals.size());
        r = rand() % evals.size();
        board.change(evals[r].move);
        prev_move = evals[r].move;
        Board cpy_board2;
        cpy_board2.copy(&board);
        game.boards.push_back(cpy_board2);

        move_cnt++;
    }

    return game;
}

void export_pgn(const char *path)
{
    auto pgn_games = PGN::import(path);

    FILE *train_data_file = fopen("temp/train.data", "wb");
    FILE *train_lbl_file = fopen("temp/train.lbl", "wb");
    FILE *test_data_file = fopen("temp/test.data", "wb");
    FILE *test_lbl_file = fopen("temp/test.lbl", "wb");

    float data_buf[CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 1];
    float lbl_buf;

    for (auto pgn_game : pgn_games)
    {
        Board board;
        bool white = true;

        int game_move_cnt = 0;

        for (auto move_str : pgn_game->move_strs)
        {
            auto move = board.change(move_str, white);
            white = !white;

            if (game_move_cnt > 6)
            {
                board.one_hot_encode(data_buf);
                data_buf[CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT] = white ? 1.0f : -1.0f;

                lbl_buf = (float)pgn_game->lbl;

                if (rand() % 20 == 0)
                {
                    fwrite(data_buf, sizeof(data_buf), 1, test_data_file);
                    fwrite(&lbl_buf, sizeof(lbl_buf), 1, test_lbl_file);
                }
                else
                {
                    fwrite(data_buf, sizeof(data_buf), 1, train_data_file);
                    fwrite(&lbl_buf, sizeof(lbl_buf), 1, train_lbl_file);
                }
            }

            game_move_cnt++;
        }

        delete pgn_game;
    }

    fclose(train_data_file);
    fclose(train_lbl_file);
    fclose(test_data_file);
    fclose(test_lbl_file);
}

void export_pgn2(const char *path)
{
    auto pgn_games = PGN::import(path);

    FILE *train_data_file = fopen("temp/train2.data", "wb");
    FILE *train_lbl_file = fopen("temp/train2.lbl", "wb");

    float data_buf[CHESS_BOARD_CHANNEL_CNT * 2 * CHESS_ROW_CNT * CHESS_COL_CNT];
    float lbl_buf;

    int g = 0;

    for (auto pgn_game : pgn_games)
    {
        Board board;
        bool white = true;

        int game_move_cnt = 0;

        for (auto move_str : pgn_game->move_strs)
        {
            auto move = board.change(move_str, white);
            white = !white;

            if (game_move_cnt > 6)
            {
                board.one_hot_encode_w_moves(data_buf, white);
                lbl_buf = (float)pgn_game->lbl;

                fwrite(data_buf, sizeof(data_buf), 1, train_data_file);
                fwrite(&lbl_buf, sizeof(lbl_buf), 1, train_lbl_file);
            }

            game_move_cnt++;
        }

        g++;

        delete pgn_game;
    }

    fclose(train_data_file);
    fclose(train_lbl_file);
}

struct Batch
{
    zero::core::Tensor *x;
    zero::core::Tensor *y;
};

std::vector<Batch> get_dataset(const char *data_path, const char *lbl_path, int batch_size)
{
    int oh_board_len = CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 1;
    int oh_board_size = oh_board_len * sizeof(float);

    long long data_file_size = FileUtils::get_file_size(data_path);
    size_t data_cnt = data_file_size / oh_board_size;

    std::vector<Batch> batches;

    FILE *data_file = fopen(data_path, "rb");
    FILE *lbl_file = fopen(lbl_path, "rb");

    float *data_buf = (float *)malloc(data_file_size);
    fread(data_buf, 1, (data_file_size), data_file);

    float *lbl_buf = (float *)malloc(sizeof(float) * data_cnt);
    fread(lbl_buf, 1, (sizeof(float) * data_cnt), lbl_file);

    fclose(data_file);
    fclose(lbl_file);

    for (int i = 0; i < data_cnt / batch_size; i++)
    {
        auto x = Tensor::from_data(Shape(batch_size, oh_board_len), &data_buf[i * batch_size * oh_board_len]);
        auto y = Tensor::from_data(Shape(batch_size, 1), &lbl_buf[i * batch_size]);

        batches.push_back({x, y});
    }

    free(data_buf);
    free(lbl_buf);

    return batches;
}

std::vector<Batch> get_dataset2(const char *data_path, const char *lbl_path, int batch_size)
{
    int oh_board_len = CHESS_BOARD_CHANNEL_CNT * 2 * CHESS_ROW_CNT * CHESS_COL_CNT;
    int oh_board_size = oh_board_len * sizeof(float);

    long long data_file_size = FileUtils::get_file_size(data_path);
    size_t data_cnt = data_file_size / oh_board_size;

    std::vector<Batch> batches;

    FILE *data_file = fopen(data_path, "rb");
    FILE *lbl_file = fopen(lbl_path, "rb");

    float *data_buf = (float *)malloc(data_file_size);
    fread(data_buf, 1, (data_file_size), data_file);

    float *lbl_buf = (float *)malloc(sizeof(float) * data_cnt);
    fread(lbl_buf, 1, (sizeof(float) * data_cnt), lbl_file);

    fclose(data_file);
    fclose(lbl_file);

    for (int i = 0; i < data_cnt / batch_size; i++)
    {
        auto x = Tensor::from_data(Shape(batch_size, CHESS_BOARD_CHANNEL_CNT * 2, CHESS_ROW_CNT, CHESS_COL_CNT), &data_buf[i * batch_size * oh_board_len]);
        auto y = Tensor::from_data(Shape(batch_size, 1), &lbl_buf[i * batch_size]);

        batches.push_back({x, y});
    }

    free(data_buf);
    free(lbl_buf);

    return batches;
}

void train_n_test(Model *model, int epochs, std::vector<Batch> *train_ds, std::vector<Batch> *test_ds)
{
    int train_batch_cnt = train_ds->size();
    int test_batch_cnt = test_ds->size();

    // Train:
    {
        FILE *train_csv = fopen("temp/train.csv", "w");
        fprintf(train_csv, "epoch,batch,loss,accuracy\n");

        bool quit = false;

        for (int epoch = 0; epoch < epochs; epoch++)
        {
            for (int batch_idx = 0; batch_idx < train_batch_cnt; batch_idx++)
            {
                auto batch = &train_ds->at(batch_idx);
                auto x = batch->x;
                auto y = batch->y;

                auto p = model->forward(x);

                float loss = model->loss(p, y);
                float acc = model->accuracy(p, y, chess_tanh_accuracy_fn);
                fprintf(train_csv, "%d,%d,%f,%f\n", epoch, batch_idx, loss, acc);

                model->backward(p, y);
                model->step();

                delete p;

                x->to_cpu();
                y->to_cpu();

                if (_kbhit())
                {
                    if (_getch() == 'q')
                    {
                        quit = true;
                        break;
                    }
                }
            }

            if (quit)
            {
                break;
            }
        }

        fclose(train_csv);
    }

    // Test:
    {
        float loss = 0.0f;
        float acc = 0.0f;

        for (int batch_idx = 0; batch_idx < test_batch_cnt; batch_idx++)
        {
            auto batch = &test_ds->at(batch_idx);
            auto x = batch->x;
            auto y = batch->y;

            auto p = model->forward(x);

            loss += model->loss(p, y);
            acc += model->accuracy(p, y, chess_tanh_accuracy_fn);

            if (batch_idx < 3)
            {
                p->print();
                y->print();
            }

            delete p;

            x->to_cpu();
            y->to_cpu();
        }

        float test_acc_pct = (acc / (float)test_batch_cnt) * 100.0f;

        printf("TEST LOSS: %f\tTEST ACCURACY: %f%%\n",
               (loss / (float)test_batch_cnt),
               test_acc_pct);
    }
}

void train2(Model *model, int epochs, int batch_size)
{
    const char *data_path = "temp/train2.data";
    const char *lbl_path = "temp/train2.lbl";

    int input_size = (CHESS_BOARD_CHANNEL_CNT * 2 * CHESS_ROW_CNT * CHESS_COL_CNT) * sizeof(float);

    long long data_file_size = FileUtils::get_file_size(data_path);
    size_t data_cnt = data_file_size / input_size;

    int batch_cnt = data_cnt / batch_size;

    FILE *data_file = fopen(data_path, "rb");
    FILE *lbl_file = fopen(lbl_path, "rb");

    // Train:
    {
        FILE *train_csv = fopen("temp/train.csv", "w");
        fprintf(train_csv, "epoch,batch,loss,accuracy\n");

        bool quit = false;

        auto x = Tensor::zeros(false, Shape(batch_size, CHESS_BOARD_CHANNEL_CNT * 2, CHESS_ROW_CNT, CHESS_COL_CNT));
        auto y = Tensor::zeros(false, Shape(batch_size, 1));

        for (int epoch = 0; epoch < epochs; epoch++)
        {
            for (int batch_idx = 0; batch_idx < batch_cnt; batch_idx++)
            {
                x->to_cpu();
                y->to_cpu();

                fread(x->data(), 1, (input_size * batch_size), data_file);
                fread(y->data(), 1, (sizeof(float) * batch_size), lbl_file);

                auto p = model->forward(x);

                if (batch_idx % 100 == 0)
                {
                    float loss = model->loss(p, y);
                    float acc = model->accuracy(p, y, chess_tanh_accuracy_fn);
                    fprintf(train_csv, "%d,%d,%f,%f\n", epoch, batch_idx, loss, acc);
                }

                model->backward(p, y);
                model->step();

                if (batch_idx == 0)
                {
                    p->print();
                    y->print();
                    for (int i = 0; i < batch_size; i++)
                    {
                        p->set_val(i, p->get_val(i) - y->get_val(i));
                    }
                    p->print();
                }

                delete p;

                if (_kbhit())
                {
                    if (_getch() == 'q')
                    {
                        quit = true;
                        break;
                    }
                }
            }

            fseek(data_file, 0, SEEK_SET);
            fseek(lbl_file, 0, SEEK_SET);

            if (quit)
            {
                break;
            }
        }

        delete x;
        delete y;

        fclose(train_csv);
    }

    fclose(data_file);
    fclose(lbl_file);
}

void grad_tests()
{
    auto test_ds = get_dataset("temp/test.data", "temp/test.lbl", 1);

    auto x = test_ds[0].x;
    auto y = test_ds[0].y;

    Shape x_shape = x->shape();
    Shape y_shape = y->shape();

    {
        auto model = new Model();
        model->set_initializer(new ChessInitializer());
        model->linear(x_shape, 32, new Tanh());
        model->linear(32, new Tanh());
        model->linear(32, new Tanh());
        model->linear(y_shape, new Tanh());
        model->set_loss(new MSE());

        model->summarize();
        model->validate_gradients(x, y, false);

        delete model;
    }
}

void compare_models(int epochs)
{
    auto train_ds = get_dataset("temp/train.data", "temp/train.lbl", 128);
    auto test_ds = get_dataset("temp/test.data", "temp/test.lbl", 128);

    Shape x_shape = train_ds[0].x->shape();
    Shape y_shape = train_ds[0].y->shape();

    {
        auto model = new Model(new ChessInitializer());

        model->linear(x_shape, 2048, new Tanh());
        model->linear(128, new Tanh());
        model->linear(y_shape, new Tanh());

        model->set_loss(new MSE());
        model->set_optimizer(new ChessOptimizer(model->parameters(), 0.01f, ZERO_NN_BETA_1));

        model->summarize();

        train_n_test(model, epochs, &train_ds, &test_ds);

        delete model;
    }

    for (auto batch : train_ds)
    {
        delete batch.x;
        delete batch.y;
    }

    for (auto batch : test_ds)
    {
        delete batch.x;
        delete batch.y;
    }
}

void compare_models2(int epochs, int batch_size)
{
    Shape x_shape(batch_size, CHESS_BOARD_CHANNEL_CNT * 2, CHESS_ROW_CNT, CHESS_COL_CNT);
    Shape y_shape(batch_size, 1);

    // {
    //     auto model = new Model(new ChessInitializer());

    //     model->hadamard_product(x_shape, 1, new Tanh());
    //     model->linear(512, new Tanh());
    //     model->linear(512, new Tanh());
    //     model->linear(64, new Tanh());
    //     model->linear(y_shape, new Tanh());

    //     model->set_loss(new MSE());
    //     model->set_optimizer(new ChessOptimizer(model->parameters(), 0.01f, ZERO_NN_BETA_1));

    //     model->summarize();

    //     train2(model, epochs, batch_size);

    //     delete model;
    // }

    {
        auto model = new Model(new ChessInitializer());

        model->linear(x_shape, 1024, new Tanh());
        model->linear(512, new Tanh());
        model->linear(64, new Tanh());
        model->linear(y_shape, new Tanh());

        model->set_loss(new MSE());
        model->set_optimizer(new ChessOptimizer(model->parameters(), 0.001f, ZERO_NN_BETA_1));

        model->summarize();

        train2(model, epochs, batch_size);

        delete model;
    }
}

int main()
{
    srand(time(NULL));

    // export_pgn("data/data.pgn");

    // grad_tests();

    // compare_models(5);

    export_pgn2("data/all.pgn");

    // compare_models2(10, 128);

    return 0;
}