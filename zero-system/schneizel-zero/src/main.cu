#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>

#include <map>

#include <zero/mod.cuh>

#include "chess.h"

using namespace zero::core;
using namespace zero::nn;

using namespace chess;

#define CHESS_BOARD_CHANNEL_CNT 12

void one_hot_encode_chess_board_data(const char *board_data, float *out)
{
    memset(out, 0, sizeof(float) * CHESS_BOARD_CHANNEL_CNT * CHESS_BOARD_LEN);
    for (int c = 0; c < CHESS_BOARD_CHANNEL_CNT; c++)
    {
        for (int i = 0; i < CHESS_ROW_CNT; i++)
        {
            for (int j = 0; j < CHESS_COL_CNT; j++)
            {
                int channel_offset = (c * CHESS_BOARD_LEN);
                int square = (i * CHESS_COL_CNT) + j;
                int out_idx = channel_offset + square;

                switch (c)
                {
                case 0:
                    if (board_data[square] == CHESS_WP)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 1:
                    if (board_data[square] == CHESS_WN)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 2:
                    if (board_data[square] == CHESS_WB)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 3:
                    if (board_data[square] == CHESS_WR)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 4:
                    if (board_data[square] == CHESS_WQ)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 5:
                    if (board_data[square] == CHESS_WK)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 6:
                    if (board_data[square] == CHESS_BP)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 7:
                    if (board_data[square] == CHESS_BN)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 8:
                    if (board_data[square] == CHESS_BB)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 9:
                    if (board_data[square] == CHESS_BR)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 10:
                    if (board_data[square] == CHESS_BQ)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                case 11:
                    if (board_data[square] == CHESS_BK)
                    {
                        out[out_idx] = 1.0f;
                    }
                    break;
                default:
                    break;
                }
            }
        }
    }
}

void play(bool white, int depth)
{
    Board board;
    Move prev_move;

    OpeningEngine opening_engine("data/openings.data");
    bool opening_stage = true;

    int move_cnt = 0;

    while (true)
    {
        printf("\nWHITE TURN\n");
        if (move_cnt == 0)
        {
            board.print();
        }
        else
        {
            board.print(prev_move);
        }

        if (board.is_checkmate(false))
        {
            printf("WHITE CHECKMATED!\n");
            break;
        }
        else if (!board.has_moves(true))
        {
            printf("WHITE STALEMATED!\n");
            break;
        }

        if (white)
        {
            do
            {
                std::string move_str;
                printf("Enter Move: ");
                std::cin >> move_str;
                prev_move = board.change(move_str, true);
            } while (!Move::is_valid(&prev_move));
        }
        else
        {
            if (move_cnt == 0)
            {
                // Default opening if white.
                prev_move = board.change("e4", true);
            }
            else
            {
                if (opening_stage)
                {
                    std::string move_str = opening_engine.next_move(&board, move_cnt);

                    if (move_str.empty())
                    {
                        printf("End of opening stage\n");
                        opening_stage = false;
                    }
                    else
                    {
                        prev_move = board.change(move_str, true);
                    }
                }

                if (!opening_stage)
                {
                    auto evals = board.minimax_alphabeta(true, depth, 7, 10);
                    int r = rand() % evals.size();
                    board.change(evals[r].move);
                    prev_move = evals[r].move;
                    printf("Ties: %d\n", evals.size());
                }
            }
        }

        move_cnt++;

        printf("\nBLACK TURN\n");
        board.print(prev_move);

        if (board.is_checkmate(true))
        {
            printf("BLACK CHECKMATED!\n");
            break;
        }
        else if (!board.has_moves(false))
        {
            printf("BLACK STALEMATED!\n");
            break;
        }

        if (!white)
        {
            do
            {
                std::string move_str;
                printf("Enter Move: ");
                std::cin >> move_str;
                prev_move = board.change(move_str, false);
            } while (!Move::is_valid(&prev_move));
        }
        else
        {

            if (opening_stage)
            {
                std::string move_str = opening_engine.next_move(&board, move_cnt);

                if (move_str.empty())
                {
                    printf("End of opening stage\n");
                    opening_stage = false;
                }
                else
                {
                    prev_move = board.change(move_str, false);
                }
            }

            if (!opening_stage)
            {
                auto evals = board.minimax_alphabeta(false, depth, 7, 10);
                int r = rand() % evals.size();
                board.change(evals[r].move);
                prev_move = evals[r].move;
                printf("Ties: %d\n", evals.size());
            }
        }

        move_cnt++;
    }
}

void selfplay(int depth, Model *model)
{
    Board board;
    Move prev_move;

    OpeningEngine opening_engine("data/openings.data");
    bool opening_stage = true;

    int move_cnt = 0;

    int x_size = (CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 2);
    auto x = Tensor::zeros(false, Shape(1, x_size));

    while (true)
    {
        printf("\nWHITE TURN\n");
        if (move_cnt == 0)
        {
            board.print();
        }
        else
        {
            board.print(prev_move);
        }

        if (board.is_checkmate(false))
        {
            printf("WHITE CHECKMATED!\n");
            break;
        }
        else if (!board.has_moves(true))
        {
            printf("WHITE STALEMATED!\n");
            break;
        }

        // White:
        {
            printf("MATERIAL EVALUATION: %d\n", board.evaluate_material());

            if (move_cnt == 0)
            {
                // Default opening if white.
                prev_move = board.change("e4", true);

                // std::string move_str = opening_engine.next_move(&board, move_cnt);
                // prev_move = board.change(move_str, true);
            }
            else
            {
                if (opening_stage)
                {
                    std::string move_str = opening_engine.next_move(&board, move_cnt);

                    if (move_str.empty())
                    {
                        printf("End of opening stage\n");
                        opening_stage = false;
                    }
                    else
                    {
                        prev_move = board.change(move_str, true);
                    }
                }

                if (!opening_stage)
                {
                    auto evals = board.minimax_alphabeta(true, depth, 9, 6);

                    int max_eval_idx = 0;

                    {
                        x->to_cpu();
                        one_hot_encode_chess_board_data(board.get_data(), x->data());
                        x->data()[(CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT)] = 1.0f;
                        x->data()[(CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 1)] = 0.0f;
                        auto p = model->forward(x);

                        float max_val = 0.0f;

                        for (int eval_idx = 0; eval_idx < evals.size(); eval_idx++)
                        {
                            auto move = evals[eval_idx].move;

                            float p_val = p->get_val(move.src_square);

                            // Incentivize castling and disincentivize moving king.
                            if (board.get_king_square(true) == move.src_square)
                            {
                                int src_dst_diff = abs(move.src_square - move.dst_square);
                                if (src_dst_diff == 2 || src_dst_diff == 3)
                                {
                                    p_val = 1.0f;
                                }
                                else
                                {
                                    p_val = 0.01f;
                                }
                            }

                            if (p_val >= 0.01f)
                            {
                                printf("Src: %d\tDst: %d\tPiece: %c\tVal: %f\n", move.src_square, move.dst_square, board.get_piece(move.src_square), p_val);
                            }

                            if (p_val > max_val)
                            {
                                max_eval_idx = eval_idx;
                                max_val = p_val;
                            }
                        }

                        p->reshape(Shape(1, CHESS_ROW_CNT, CHESS_COL_CNT));
                        p->print();
                        delete p;
                    }

                    board.change(evals[max_eval_idx].move);
                    prev_move = evals[max_eval_idx].move;
                    printf("Ties: %d\n", evals.size());
                }
            }
        }

        move_cnt++;

        printf("\nBLACK TURN\n");
        board.print(prev_move);

        if (board.is_checkmate(true))
        {
            printf("BLACK CHECKMATED!\n");
            break;
        }
        else if (!board.has_moves(false))
        {
            printf("BLACK STALEMATED!\n");
            break;
        }

        // Black:
        {
            printf("MATERIAL EVALUATION: %d\n", board.evaluate_material());

            if (opening_stage)
            {
                std::string move_str = opening_engine.next_move(&board, move_cnt);

                if (move_str.empty())
                {
                    printf("End of opening stage\n");
                    opening_stage = false;
                }
                else
                {
                    prev_move = board.change(move_str, false);
                }
            }

            if (!opening_stage)
            {
                auto evals = board.minimax_alphabeta(false, depth, 7, 10);
                int r = rand() % evals.size();
                board.change(evals[r].move);
                prev_move = evals[r].move;
                printf("Ties: %d\n", evals.size());
            }
        }

        move_cnt++;
    }

    delete x;
}

void export_pgn(const char *path)
{
    auto pgn_games = PGN::import(path, FileUtils::get_file_size(path));

    FILE *train_data_file = fopen("temp/train.data", "wb");
    FILE *train_lbl_file = fopen("temp/train.lbl", "wb");

    char data_buf[CHESS_BOARD_LEN + 1];
    int lbl_buf;

    int game_cnt = 0;

    long long move_cnt = 0;

    for (auto pgn_game : pgn_games)
    {
        Board board;
        bool white = true;

        int game_move_cnt = 0;

        for (auto move_str : pgn_game->move_strs)
        {
            memset(data_buf, 0, sizeof(data_buf));
            memcpy(data_buf, board.get_data(), sizeof(char) * CHESS_BOARD_LEN);

            auto move = board.change(move_str, white);

            if (!Move::is_valid(&move))
            {
                printf("Quitting game %d on move %d\n", game_cnt, game_move_cnt);
                break;
            }

            // Skip openings.
            if (game_move_cnt >= CHESS_OPENING_MOVE_CNT)
            {
                if (white)
                {
                    data_buf[CHESS_BOARD_LEN] = 'w';
                }
                else
                {
                    data_buf[CHESS_BOARD_LEN] = 'b';
                }

                lbl_buf = move.src_square;

                fwrite(data_buf, sizeof(data_buf), 1, train_data_file);
                fwrite(&lbl_buf, sizeof(lbl_buf), 1, train_lbl_file);

                move_cnt++;
            }

            white = !white;

            game_move_cnt++;
        }

        game_cnt++;

        if (game_cnt % 1000 == 0)
        {
            printf("Game: %d\tMoves: %ld\n", game_cnt, move_cnt);
        }

        delete pgn_game;
    }

    printf("Game: %d\tMoves: %ld\n", game_cnt, move_cnt);

    fclose(train_data_file);
    fclose(train_lbl_file);
}

int chess_classification_accuracy_fn(Tensor *p, Tensor *y, int batch_size)
{
    int correct_cnt = 0;

    int output_cnt = p->dims_size() / batch_size;

    for (int i = 0; i < batch_size; i++)
    {
        float max_val = y->get_val(i * output_cnt + 0);
        int max_idx = 0;
        for (int j = 1; j < output_cnt; j++)
        {
            float val = y->get_val(i * output_cnt + j);
            if (val > max_val)
            {
                max_val = val;
                max_idx = j;
            }
        }

        if (p->get_val(i * output_cnt + max_idx) >= 0.95f)
        {
            correct_cnt++;
        }
    }

    return correct_cnt;
}

void train(Model *model, int epochs, int batch_size)
{
    const char *data_path = "temp/train.data";
    const char *lbl_path = "temp/train.lbl";

    int input_size = CHESS_BOARD_LEN + 1;
    int x_size = (CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 2);

    long long data_file_size = FileUtils::get_file_size(data_path);
    size_t data_cnt = data_file_size / input_size;

    int batch_cnt = data_cnt / batch_size;

    FILE *data_file = fopen(data_path, "rb");
    FILE *lbl_file = fopen(lbl_path, "rb");

    // Train:
    {
        FILE *train_csv = fopen("temp/train.csv", "w");
        fprintf(train_csv, "epoch,batch,loss,accuracy\n");

        bool quit = false;

        auto x = Tensor::zeros(false, Shape(batch_size, x_size));
        auto y = Tensor::zeros(false, Shape(batch_size, 1));

        char *data_buf = (char *)malloc(sizeof(char) * batch_size * input_size);
        int *lbl_buf = (int *)malloc(sizeof(int) * batch_size);

        for (int epoch = 0; epoch < epochs; epoch++)
        {
            for (int batch_idx = 0; batch_idx < batch_cnt; batch_idx++)
            {
                x->zeros();
                y->zeros();

                x->to_cpu();
                y->to_cpu();

                fread(data_buf, 1, (input_size * batch_size), data_file);
                fread(lbl_buf, 1, (sizeof(int) * batch_size), lbl_file);

                for (int i = 0; i < batch_size; i++)
                {
                    one_hot_encode_chess_board_data(&data_buf[i * input_size], &x->data()[i * x_size]);
                    if (data_buf[i * input_size + CHESS_BOARD_LEN] == 'w')
                    {
                        x->data()[(i * x_size) + (CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT)] = 1.0f;
                    }
                    else
                    {
                        x->data()[(i * x_size) + (CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 1)] = 1.0f;
                    }

                    y->data()[i] = (float)lbl_buf[i];
                }

                auto oh_y = Tensor::one_hot(y, CHESS_BOARD_LEN - 1);

                auto p = model->forward(x);

                if (batch_idx % 100 == 0)
                {
                    float loss = model->loss(p, oh_y);
                    float acc = model->accuracy(p, oh_y, chess_classification_accuracy_fn);
                    fprintf(train_csv, "%d,%d,%f,%f\n", epoch, batch_idx, loss, acc);
                }

                model->backward(p, oh_y);
                model->step();

                if (batch_idx == batch_cnt - 1)
                {
                    y->print();
                    p->print();
                }

                delete p;
                delete oh_y;

                if (_kbhit())
                {
                    if (_getch() == 'q')
                    {
                        quit = true;
                        break;
                    }
                }
            }

            fseek(data_file, 0, SEEK_SET);
            fseek(lbl_file, 0, SEEK_SET);

            if (quit)
            {
                break;
            }
        }

        delete x;
        delete y;

        free(data_buf);
        free(lbl_buf);

        fclose(train_csv);
    }

    fclose(data_file);
    fclose(lbl_file);
}

void compare_models(int epochs, int batch_size)
{
    Shape x_shape(batch_size, CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 2);
    Shape y_shape(batch_size, CHESS_BOARD_LEN);

    {
        auto model = new Model(new Xavier());

        model->linear(x_shape, 1024, new ReLU());
        model->linear(1024, new ReLU());
        model->linear(512, new ReLU());
        model->linear(512, new ReLU());
        model->linear(128, new ReLU());
        model->linear(y_shape, new Sigmoid());

        model->set_loss(new CrossEntropy());
        model->set_optimizer(new SGDMomentum(model->parameters(), 0.01f, ZERO_NN_BETA_1));

        model->summarize();

        train(model, epochs, batch_size);

        model->save_parameters("temp/model.nn");

        delete model;
    }
}

void cont_training(int epochs, int batch_size)
{
    Shape x_shape(batch_size, CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 2);
    Shape y_shape(batch_size, CHESS_BOARD_LEN);

    {
        auto model = new Model(new Xavier());

        model->linear(x_shape, 1024, new ReLU());
        model->linear(1024, new ReLU());
        model->linear(512, new ReLU());
        model->linear(512, new ReLU());
        model->linear(128, new ReLU());
        model->linear(y_shape, new Sigmoid());

        model->set_loss(new CrossEntropy());
        model->set_optimizer(new SGDMomentum(model->parameters(), 0.001f, ZERO_NN_BETA_1));

        model->load_parameters("temp/model.nn");

        model->summarize();

        train(model, epochs, batch_size);

        // model->save_parameters("temp/model.nn");

        delete model;
    }
}

int main()
{
    srand(time(NULL));

    // export_pgn("data/all.pgn");

    // compare_models(10, 64);

    // cont_training(10, 64);

    Shape x_shape(1, CHESS_BOARD_CHANNEL_CNT * CHESS_ROW_CNT * CHESS_COL_CNT + 2);
    Shape y_shape(1, CHESS_BOARD_LEN);

    auto model = new Model(new Xavier());
    {
        model->linear(x_shape, 1024, new ReLU());
        model->linear(1024, new ReLU());
        model->linear(512, new ReLU());
        model->linear(512, new ReLU());
        model->linear(128, new ReLU());
        model->linear(y_shape, new Sigmoid());

        model->set_loss(new CrossEntropy());
        model->set_optimizer(new SGDMomentum(model->parameters(), 0.001f, ZERO_NN_BETA_1));

        model->load_parameters("temp/model.nn");
    }

    selfplay(3, model);

    delete model;

    return 0;
}