#include "model.cuh"

Model::Model()
{
}

Model::~Model()
{
    for (Layer *lyr : this->lyrs_)
    {
        delete lyr;
    }

    delete this->loss_;
}

ArrayNd *Model::forward(ArrayNd *x)
{
    x->to_cuda();

    int batch_size = x->dims().dim(0);

    this->lyrs_[0]->set_n(x);

    int lst_lyr_idx = this->lyrs_.size() - 1;

    for (int i = 0; i < lst_lyr_idx; i++)
    {
        Layer *lyr = this->lyrs_[i];
        Layer *nxt_lyr = this->lyrs_[i + 1];

        lyr->forward(nxt_lyr->n());
    }

    Layer *lst_lyr = this->lyrs_[lst_lyr_idx];

    ArrayNd *p = new ArrayNd(true, lst_lyr->n()->dims());
    lst_lyr->forward(p);

    return p;
}

void Model::backward(ArrayNd *p, ArrayNd *y)
{
    y->to_cuda();

    ArrayNd *dl = this->loss_->derive(p, y);

    int lst_lyr_idx = this->lyrs_.size() - 1;
    for (int i = lst_lyr_idx; i >= 0; i--)
    {
        Layer *lyr = this->lyrs_[i];
        dl = lyr->backward(dl);
    }

    delete dl;
}

float Model::loss(ArrayNd *p, ArrayNd *y)
{
    y->to_cuda();

    float loss_val = 0.0f;
    float *d_loss_val;

    hipMalloc(&d_loss_val, sizeof(float));
    hipMemset(d_loss_val, 0, sizeof(float));

    this->loss_->evaluate(p, y, d_loss_val);

    hipMemcpy(&loss_val, d_loss_val, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_loss_val);

    return loss_val;
}

void Model::step()
{
}