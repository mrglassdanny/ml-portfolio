#include "hip/hip_runtime.h"
#include "layer.cuh"

#define DEFAULT_BATCH_SIZE 1

using namespace nn::layer;

__global__ void k_linear_matmul_w_bias(float *in, float *w, float *out, float *b,
                                       int in_col_cnt, int out_row_cnt, int out_col_cnt)
{
    int out_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_col_idx < out_col_cnt && out_row_idx < out_row_cnt)
    {
        int out_elem_idx = out_row_idx * out_col_cnt + out_col_idx;
        int in_row_idx = out_row_idx;
        int w_col_idx = out_col_idx;

        for (int in_col_idx = 0; in_col_idx < in_col_cnt; in_col_idx++)
        {
            int w_row_idx = in_col_idx;
            out[out_elem_idx] += (in[in_row_idx * in_col_cnt + in_col_idx] * w[w_row_idx * out_col_cnt + w_col_idx]);
        }

        out[out_elem_idx] += b[w_col_idx];
    }
}

__global__ void k_linear_inc_param_derivatives(float *in, float *n, float *w, float *b, float *dw, float *db,
                                               int in_row_cnt, int in_col_cnt, int n_col_cnt, int w_row_cnt, int w_col_cnt)
{
    int w_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int w_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (w_col_idx < w_col_cnt && w_row_idx < w_row_cnt)
    {
        int w_elem_idx = w_row_idx * w_col_cnt + w_col_idx;
        int n_col_idx = w_row_idx;
        int in_col_idx = w_col_idx;

        for (int i = 0; i < in_row_cnt; i++)
        {
            dw[w_elem_idx] += (in[i * in_col_cnt + in_col_idx] * n[i * n_col_cnt + n_col_idx]);

            if (w_row_idx == 0)
            {
                int b_elem_idx = w_col_idx;
                db[b_elem_idx] += in[i * in_col_cnt + in_col_idx];
            }
        }
    }
}

__global__ void k_linear_agg_derivatives(float *in, float *w, float *out, int in_col_cnt, int w_col_cnt, int out_row_cnt, int out_col_cnt)
{
    int out_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_col_idx < out_col_cnt && out_row_idx < out_row_cnt)
    {
        int out_elem_idx = out_row_idx * w_col_cnt + out_col_idx;
        int in_row_idx = out_row_idx;
        int w_row_idx = out_col_idx;

        for (int in_col_idx = 0; in_col_idx < in_col_cnt; in_col_idx++)
        {
            int w_col_idx = in_col_idx;
            out[out_elem_idx] += (in[in_row_idx * in_col_cnt + in_col_idx] * w[w_row_idx * w_col_cnt + w_col_idx]);
        }
    }
}

__global__ void k_conv2d_pad()
{
}

__device__ void d_conv2d_convolve(float *in, float *w, float *out, int channel_cnt, int in_row_cnt, int in_col_cnt, int in_cnt,
                                  int filter_row_cnt, int filter_col_cnt, int w_cnt, int out_row_cnt, int out_col_cnt,
                                  int stride_row_cnt, int stride_col_cnt)
{
    for (int i = 0; i < channel_cnt; i++)
    {
        for (int j = 0; j < out_row_cnt; j++)
        {
            for (int k = 0; k < out_col_cnt; k++)
            {
                int out_elem_idx = j * out_col_cnt + k;
                int in_row_offset = j + stride_row_cnt;
                int in_col_offset = k + stride_col_cnt;

                for (int l = 0; l < filter_row_cnt; l++)
                {
                    for (int m = 0; m < filter_col_cnt; m++)
                    {
                        int in_elem_idx = (i * in_cnt) + (l + in_row_offset) * filter_col_cnt + (m + in_col_offset);
                        int w_elem_idx = (i * w_cnt) + (l * filter_col_cnt + m);
                        out[out_elem_idx] += (in[in_elem_idx] * w[w_elem_idx]);
                    }
                }
            }
        }
    }
}

__global__ void k_conv2d_evaluate(float *in, float *w, float *out, int batch_size, int channel_cnt, int in_row_cnt, int in_col_cnt,
                                  int filter_cnt, int filter_row_cnt, int filter_col_cnt, int out_row_cnt, int out_col_cnt,
                                  int stride_row_cnt, int stride_col_cnt)
{
    int filter_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (batch_idx < batch_size && filter_idx < filter_cnt)
    {
        int in_cnt = in_row_cnt * in_col_cnt;
        int w_cnt = filter_row_cnt * filter_col_cnt;
        int out_cnt = out_row_cnt * out_col_cnt;

        float *l_in = &in[(batch_idx * channel_cnt * in_cnt)];
        float *l_w = &w[(filter_idx * channel_cnt * w_cnt)];
        float *l_out = &out[(batch_idx * filter_cnt * out_cnt)];

        d_conv2d_convolve(l_in, l_w, l_out, channel_cnt, in_row_cnt, in_col_cnt, in_cnt,
                          filter_row_cnt, filter_col_cnt, w_cnt, out_row_cnt, out_col_cnt,
                          stride_row_cnt, stride_col_cnt);
    }
}

__device__ float d_sigmoid_evaluate(float val)
{
    return (1.0f / (1.0f + exp(-val)));
}

__device__ float d_sigmoid_derive(float val)
{
    float sigmoid_val = d_sigmoid_evaluate(val);
    return (sigmoid_val) * (1.0f - sigmoid_val);
}

__global__ void k_sigmoid_evaluate(float *in, float *out, int row_cnt, int col_cnt)
{
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (col_idx < col_cnt && row_idx < row_cnt)
    {
        int elem_idx = row_idx * col_cnt + col_idx;

        out[elem_idx] = d_sigmoid_evaluate(in[elem_idx]);
    }
}

__global__ void k_sigmoid_derive(float *in, float *n, float *out, int row_cnt, int col_cnt)
{
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (col_idx < col_cnt && row_idx < row_cnt)
    {
        int elem_idx = row_idx * col_cnt + col_idx;

        out[elem_idx] = in[elem_idx] * d_sigmoid_derive(n[elem_idx]);
    }
}

Layer::~Layer()
{
    delete this->n_;
}

int Layer::batch_size()
{
    return this->n_->shape()[0];
}

void Layer::lock_batch_size(int batch_size)
{
    this->n_->change_dim(0, batch_size);
}

NdArray *Layer::neurons()
{
    return this->n_;
}

void Layer::copy_neurons(NdArray *n)
{
    this->n_->copy(n);
}

Parameters::Parameters(Shape w_shape, Shape b_shape, int fan_in, int fan_out)
{
    this->w_ = NdArray::rands(true, w_shape, 0.0f, sqrt(1.0f / fan_in));
    this->b_ = NdArray::zeros(true, b_shape);
    this->dw_ = NdArray::zeros(true, w_shape);
    this->db_ = NdArray::zeros(true, b_shape);
}

Parameters::~Parameters()
{
    delete this->w_;
    delete this->b_;
    delete this->dw_;
    delete this->db_;
}

void Parameters::zero_grad()
{
    this->dw_->zeros();
    this->db_->zeros();
}

NdArray *Parameters::weights()
{
    return this->w_;
}

NdArray *Parameters::biases()
{
    return this->b_;
}

NdArray *Parameters::weight_gradients()
{
    return this->dw_;
}

NdArray *Parameters::bias_gradients()
{
    return this->db_;
}

Learnable::~Learnable()
{
    delete params_;
}

Parameters *Learnable::parameters()
{
    return this->params_;
}

Linear::Linear(int in_cnt, int out_cnt)
{
    this->n_ = new NdArray(true, Shape(DEFAULT_BATCH_SIZE, in_cnt));
    this->params_ = new Parameters(Shape(in_cnt, out_cnt), Shape(out_cnt), in_cnt, out_cnt);
}

void Linear::evaluate(NdArray *out)
{
    out->zeros();

    int grid_row_cnt = (out->shape()[0] / THREADS_PER_BLOCK) + 1;
    int grid_col_cnt = (out->shape()[1] / THREADS_PER_BLOCK) + 1;

    dim3 grid_dims(grid_col_cnt, grid_row_cnt);
    dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    NdArray *n = this->n_;
    NdArray *w = this->params_->weights();
    NdArray *b = this->params_->biases();

    k_linear_matmul_w_bias<<<grid_dims, block_dims>>>(n->data(), w->data(), out->data(), b->data(),
                                                      n->shape()[1], out->shape()[0], out->shape()[1]);
}

NdArray *Linear::derive(NdArray *in)
{
    NdArray *n = this->n_;
    NdArray *w = this->params_->weights();
    NdArray *b = this->params_->biases();
    NdArray *dw = this->params_->weight_gradients();
    NdArray *db = this->params_->bias_gradients();

    {
        int grid_row_cnt = (w->shape()[0] / THREADS_PER_BLOCK) + 1;
        int grid_col_cnt = (w->shape()[1] / THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

        k_linear_inc_param_derivatives<<<grid_dims, block_dims>>>(in->data(), n->data(), w->data(), b->data(), dw->data(), db->data(),
                                                                  in->shape()[0], in->shape()[1], n->shape()[1], w->shape()[0], w->shape()[1]);
    }

    NdArray *out = NdArray::zeros(true, n->shape());

    {
        int grid_row_cnt = (out->shape()[0] / THREADS_PER_BLOCK) + 1;
        int grid_col_cnt = (out->shape()[1] / THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

        k_linear_agg_derivatives<<<grid_dims, block_dims>>>(in->data(), w->data(), out->data(),
                                                            in->shape()[1], w->shape()[1], out->shape()[0], out->shape()[1]);
    }

    delete in;
    return out;
}

Shape Linear::input_shape()
{
    return this->n_->shape();
}

Shape Linear::output_shape()
{
    return Shape(this->batch_size(), this->params_->weights()->shape()[1]);
}

Conv2d::Conv2d(int channel_cnt, Shape in_shape, int filter_cnt, Shape filter_shape, Shape padding_shape, Shape stride_shape)
{
    this->n_ = new NdArray(true, Shape(DEFAULT_BATCH_SIZE, in_shape));

    int out_row_cnt = (((in_shape[0] - filter_shape[0]) + (2 * padding_shape[0])) / stride_shape[0]) + 1;
    int out_col_cnt = (((in_shape[1] - filter_shape[1]) + (2 * padding_shape[1])) / stride_shape[1]) + 1;

    this->params_ = new Parameters(Shape(filter_cnt, channel_cnt, filter_shape[0], filter_shape[1]), Shape(filter_cnt, channel_cnt), in_shape[0], in_shape[1]);

    this->channel_cnt_ = channel_cnt;
    this->in_shape_ = in_shape;
    this->filter_cnt_ = filter_cnt;
    this->filter_shape_ = filter_shape;
    this->padding_shape_ = padding_shape;
    this->stride_shape_ = stride_shape;
}

void Conv2d::evaluate(NdArray *out)
{
    out->zeros();
    // this->lock_padding();

    int grid_row_cnt = (this->batch_size() / THREADS_PER_BLOCK) + 1;
    int grid_col_cnt = (this->filter_cnt_ / THREADS_PER_BLOCK) + 1;

    dim3 grid_dims(grid_col_cnt, grid_row_cnt);
    dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    NdArray *n = this->n_;
    NdArray *w = this->params_->weights();
    NdArray *b = this->params_->biases();

    int out_row_cnt = (((this->in_shape_[0] - this->filter_shape_[0]) + (2 * this->padding_shape_[0])) / this->stride_shape_[0]) + 1;
    int out_col_cnt = (((this->in_shape_[1] - this->filter_shape_[1]) + (2 * this->padding_shape_[1])) / this->stride_shape_[1]) + 1;

    k_conv2d_evaluate<<<grid_dims, block_dims>>>(n->data(), w->data(), out->data(), this->batch_size(), this->channel_cnt_, this->in_shape_[0], this->in_shape_[1],
                                                 this->filter_cnt_, this->filter_shape_[0], this->filter_shape_[1], out_row_cnt, out_col_cnt,
                                                 this->stride_shape_[0], this->stride_shape_[1]);
}

NdArray *Conv2d::derive(NdArray *in)
{
    return nullptr;
}

Shape Conv2d::input_shape()
{
    return Shape(this->batch_size(), this->channel_cnt_, this->in_shape_[0], this->in_shape_[1]);
}

Shape Conv2d::output_shape()
{
    int out_row_cnt = (((this->in_shape_[0] - this->filter_shape_[0]) + (2 * this->padding_shape_[0])) / this->stride_shape_[0]) + 1;
    int out_col_cnt = (((this->in_shape_[1] - this->filter_shape_[1]) + (2 * this->padding_shape_[1])) / this->stride_shape_[1]) + 1;

    return Shape(this->batch_size(), this->filter_cnt_, out_row_cnt, out_col_cnt);
}

Activation::Activation(int in_cnt)
{
    this->n_ = new NdArray(true, Shape(DEFAULT_BATCH_SIZE, in_cnt));
}

Shape Activation::input_shape()
{
    return this->n_->shape();
}

Shape Activation::output_shape()
{
    return this->n_->shape();
}

Sigmoid::Sigmoid(int in_cnt)
    : Activation(in_cnt)
{
}

void Sigmoid::evaluate(NdArray *out)
{
    out->zeros();

    int grid_row_cnt = (out->shape()[0] / THREADS_PER_BLOCK) + 1;
    int grid_col_cnt = (out->shape()[1] / THREADS_PER_BLOCK) + 1;

    dim3 grid_dims(grid_col_cnt, grid_row_cnt);
    dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    k_sigmoid_evaluate<<<grid_dims, block_dims>>>(this->n_->data(), out->data(), out->shape()[0], out->shape()[1]);
}

NdArray *Sigmoid::derive(NdArray *in)
{
    NdArray *out = new NdArray(true, this->n_->shape());

    int grid_row_cnt = (out->shape()[0] / THREADS_PER_BLOCK) + 1;
    int grid_col_cnt = (out->shape()[1] / THREADS_PER_BLOCK) + 1;

    dim3 grid_dims(grid_col_cnt, grid_row_cnt);
    dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    k_sigmoid_derive<<<grid_dims, block_dims>>>(in->data(), this->n_->data(), out->data(), out->shape()[0], out->shape()[1]);

    delete in;
    return out;
}
