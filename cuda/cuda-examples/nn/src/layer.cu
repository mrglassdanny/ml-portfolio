#include "hip/hip_runtime.h"
#include "layer.cuh"

using namespace layer;

__global__ void k_linear_matmul_w_bias(float *in, float *w, float *out, float *b,
                                       int in_col_cnt, int out_row_cnt, int out_col_cnt)
{
    int out_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_col_idx < out_col_cnt && out_row_idx < out_row_cnt)
    {
        int out_elem_idx = out_row_idx * out_col_cnt + out_col_idx;
        int in_row_idx = out_row_idx;
        int w_col_idx = out_col_idx;

        for (int in_col_idx = 0; in_col_idx < in_col_cnt; in_col_idx++)
        {
            int w_row_idx = in_col_idx;
            out[out_elem_idx] += (in[in_row_idx * in_col_cnt + in_col_idx] * w[w_row_idx * out_col_cnt + w_col_idx]);
        }

        out[out_elem_idx] += b[w_col_idx];
    }
}

__global__ void k_linear_inc_param_derivatives(float *in, float *n, float *w, float *b, float *dw, float *db,
                                               int in_row_cnt, int in_col_cnt, int n_col_cnt, int w_row_cnt, int w_col_cnt)
{
    int w_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int w_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (w_col_idx < w_col_cnt && w_row_idx < w_row_cnt)
    {
        int w_elem_idx = w_row_idx * w_col_cnt + w_col_idx;
        int n_col_idx = w_row_idx;
        int in_col_idx = w_col_idx;

        for (int i = 0; i < in_row_cnt; i++)
        {
            dw[w_elem_idx] += (in[i * in_col_cnt + in_col_idx] * n[i * n_col_cnt + n_col_idx]);

            if (w_row_idx == 0)
            {
                db[w_col_idx] += in[i * in_col_cnt + in_col_idx];
            }
        }
    }
}

__global__ void k_linear_agg_derivatives(float *in, float *w, float *out, int in_col_cnt, int w_col_cnt, int out_row_cnt, int out_col_cnt)
{
    int out_col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int out_row_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_col_idx < out_col_cnt && out_row_idx < out_row_cnt)
    {
        int out_elem_idx = out_row_idx * w_col_cnt + out_col_idx;
        int in_row_idx = out_row_idx;
        int w_row_idx = out_col_idx;

        for (int in_col_idx = 0; in_col_idx < in_col_cnt; in_col_idx++)
        {
            int w_col_idx = in_col_idx;
            out[out_elem_idx] += (in[in_row_idx * in_col_cnt + in_col_idx] * w[w_row_idx * w_col_cnt + w_col_idx]);
        }
    }
}

Linear::Linear(int in_cnt, int out_cnt)
{
    this->n_ = new NdArray(true, 1, in_cnt);
    this->w_ = new NdArray(true, in_cnt, out_cnt);
    this->b_ = new NdArray(true, out_cnt);
    this->dw_ = new NdArray(true, in_cnt, out_cnt);
    this->db_ = new NdArray(true, out_cnt);

    this->w_->rands(0.0f, sqrt(1.0f / in_cnt));
    this->b_->zeros();
    this->dw_->zeros();
    this->db_->zeros();
}

Linear::~Linear()
{
    delete this->n_;
    delete this->w_;
    delete this->b_;
    delete this->dw_;
    delete this->db_;
}

void Linear::forward(NdArray *out)
{
    out->zeros();

    {
        unsigned int grid_row_cnt = (this->n_->rows() / THREADS_PER_BLOCK) + 1;
        unsigned int grid_col_cnt = (out->cols() / THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

        k_linear_matmul_w_bias<<<grid_dims, block_dims>>>(this->n_->data(), this->w_->data(), out->data(), this->b_->data(),
                                                          this->n_->cols(), this->n_->rows(), out->cols());
    }
}

NdArray *Linear::backward(NdArray *in)
{
    {
        unsigned int grid_row_cnt = (this->w_->rows() / THREADS_PER_BLOCK) + 1;
        unsigned int grid_col_cnt = (this->w_->cols() / THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

        k_linear_inc_param_derivatives<<<grid_dims, block_dims>>>(in->data(), this->n_->data(), this->w_->data(), this->b_->data(), this->dw_->data(), this->db_->data(),
                                                                  in->rows(), in->cols(), this->n_->cols(), this->w_->rows(), this->w_->cols());
    }

    NdArray *out = new NdArray(true, this->n_->rows(), this->n_->cols());
    out->zeros();

    {
        unsigned int grid_row_cnt = (out->rows() / THREADS_PER_BLOCK) + 1;
        unsigned int grid_col_cnt = (out->cols() / THREADS_PER_BLOCK) + 1;

        dim3 grid_dims(grid_col_cnt, grid_row_cnt);
        dim3 block_dims(THREADS_PER_BLOCK, THREADS_PER_BLOCK);

        k_linear_agg_derivatives<<<grid_dims, block_dims>>>(in->data(), this->w_->data(), out->data(),
                                                            in->cols(), this->w_->cols(), out->rows(), out->cols());
    }

    delete in;
    return out;
}

NdArray *Linear::n()
{
    return this->n_;
}

void Linear::set_n(NdArray *n)
{
    
}

Activation::Activation(activation::Activation *a, int in_cnt)
{
    this->n_ = new NdArray(true, 1, in_cnt);
    this->a_ = a;
}

Activation::~Activation()
{
    delete this->n_;
    delete this->a_;
}

void Activation::forward(NdArray *out)
{
    out->zeros();

    this->a_->evaluate(this->n_, out);
}

NdArray *Activation::backward(NdArray *in)
{
    NdArray *out = new NdArray(true, in->rows(), in->cols());

    this->a_->derive(in, out);

    delete in;
    return out;
}

NdArray *Activation::n()
{
    return this->n_;
}

void Activation::set_n(NdArray *n)
{
    
}