#include "hip/hip_runtime.h"
#include "layer.cuh"

// Device functions:

// Kernel functions:

__global__ void k_linear_matmul_w_bias(float *in_mtx, float *w_mtx, float *out_mtx, float *b_vec,
                                       int in_col_cnt, int out_col_cnt, int out_elem_cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int out_elem_idx = tid;

    if (out_elem_idx < out_elem_cnt)
    {
        int in_row_idx = out_elem_idx / out_col_cnt;
        int w_col_idx = out_elem_idx % out_col_cnt;

        for (int in_col_idx = 0; in_col_idx < in_col_cnt; in_col_idx++)
        {
            out_mtx[out_elem_idx] += (in_mtx[in_row_idx * in_col_cnt + in_col_idx] * w_mtx[w_col_idx + (in_col_idx * out_col_cnt)]);
        }

        out_mtx[out_elem_idx] += b_vec[w_col_idx];
    }
}

// Layer:

Layer::Layer()
{
    this->n = NULL;
}

Layer::~Layer()
{
    delete this->n;
}

LinearLayer::LinearLayer(int in_cnt, int out_cnt)
{
    this->w = new Tensor(false, Dimensions(in_cnt, out_cnt));
    this->b = new Tensor(false, Dimensions(out_cnt));

    this->w->rands(0.0f, sqrt(1.0f / in_cnt));
    this->b->zeros();
}

LinearLayer::~LinearLayer()
{
    delete this->w;
    delete this->b;
}

void LinearLayer::forward(Tensor *out)
{
    out->zeros();

    int batch_size = this->n->get_dims().get_dim(0);
    int in_cnt = this->n->get_dims().get_dim(1);
    int out_cnt = out->get_dims().get_dim(1);

    {
        int num_blocks = ((batch_size * out_cnt) / THREADS_PER_BLOCK) + 1;
        k_linear_matmul_w_bias<<<num_blocks, THREADS_PER_BLOCK>>>(this->n->get_data(), this->w->get_data(), out->get_data(), this->b->get_data(), in_cnt, out_cnt, (batch_size * out_cnt));
    }
}

Tensor *LinearLayer::backward(Tensor *d_l)
{
    return NULL;
}