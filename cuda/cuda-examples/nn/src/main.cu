#include "hip/hip_runtime.h"
#include <util.cuh>
#include <tensor.cuh>

#define THREADS_PER_BLOCK 32

#define BATCH_SIZE 16
#define INPUT_SIZE 48
#define OUTPUT_SIZE 12

/*
	Example matrix multiplication:

			[ 1 2 1 ]
			[ 3 1 2 ]
			[ 3 3 1 ]

			x

			[ 1 2 ]
			[ 2 2 ]
			[ 1 3 ]

			=

			[  6  9  ]
			[  7 14  ]
			[ 10 15  ]
*/

void set_vals(float *A, float *B)
{
	A[0] = 1;
	A[1] = 2;
	A[2] = 1;
	A[3] = 3;
	A[4] = 1;
	A[5] = 2;
	A[6] = 3;
	A[7] = 3;
	A[8] = 1;

	B[0] = 1;
	B[1] = 2;
	B[2] = 2;
	B[3] = 2;
	B[4] = 1;
	B[5] = 3;
}

void matmul(float *A, float *B, float *C)
{
	for (int i = 0; i < BATCH_SIZE; i++)
	{
		for (int j = 0; j < INPUT_SIZE; j++)
		{
			for (int k = 0; k < OUTPUT_SIZE; k++)
			{
				C[i * OUTPUT_SIZE + k] += (A[i * INPUT_SIZE + j] * B[j * OUTPUT_SIZE + k]);
			}
		}
	}
}

__global__ void k_matmul_1(float *A, float *B, float *C)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < (BATCH_SIZE * OUTPUT_SIZE))
	{
		int A_idx = tid / OUTPUT_SIZE;
		int B_idx = tid % OUTPUT_SIZE;

		for (int i = 0; i < INPUT_SIZE; i++)
		{
			C[tid] += (A[A_idx * INPUT_SIZE + i] * B[B_idx + (i * OUTPUT_SIZE)]);
		}
	}
}

__global__ void k_matmul_2(float *A, float *B, float *C)
{
	__shared__ float temp[THREADS_PER_BLOCK];
	memset(temp, 0, THREADS_PER_BLOCK * sizeof(float));

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	int batch_idx = tid / (INPUT_SIZE * OUTPUT_SIZE);
	int A_idx = (tid % INPUT_SIZE) + (batch_idx * INPUT_SIZE);
	int B_idx = (tid % INPUT_SIZE) * OUTPUT_SIZE + ((tid - (batch_idx * (INPUT_SIZE * OUTPUT_SIZE))) / INPUT_SIZE);

	if (tid < (BATCH_SIZE * INPUT_SIZE * OUTPUT_SIZE))
	{
		temp[threadIdx.x] = A[A_idx] * B[B_idx];
	}

	__syncthreads();

	if (threadIdx.x == 0)
	{
		int lower_idx = tid / INPUT_SIZE;
		int upper_idx = ((tid + THREADS_PER_BLOCK) / INPUT_SIZE);

		if (INPUT_SIZE >= THREADS_PER_BLOCK)
		{
			if (lower_idx == upper_idx)
			{
				float sum = 0.0f;

				for (int i = 0; i < THREADS_PER_BLOCK; i++)
				{
					sum += temp[i];
				}

				atomicAdd(&C[lower_idx], sum);
			}
			else
			{
				float sums[2] = {0.0f, 0.0f};

				for (int i = 0; i < THREADS_PER_BLOCK; i++)
				{
					int idx = ((tid + i) / INPUT_SIZE);
					if (idx == lower_idx)
					{
						sums[0] += temp[i];
					}
					else
					{
						sums[1] += temp[i];
					}
				}

				atomicAdd(&C[lower_idx], sums[0]);
				if (upper_idx < (BATCH_SIZE * OUTPUT_SIZE))
				{
					atomicAdd(&C[upper_idx], sums[1]);
				}
			}
		}
		else
		{
			for (int i = 0; i < THREADS_PER_BLOCK; i++)
			{
				int idx = ((tid + i) / INPUT_SIZE);
				if (idx < (BATCH_SIZE * OUTPUT_SIZE))
				{
					atomicAdd(&C[idx], temp[i]);
				}
			}
		}
	}
}

int main(int argc, char **argv)
{
	StopWatch sw;

	Tensor *C1 = new Tensor(false, Dimensions(BATCH_SIZE, OUTPUT_SIZE));
	Tensor *C2 = new Tensor(true, Dimensions(BATCH_SIZE, OUTPUT_SIZE));
	Tensor *C3 = new Tensor(true, Dimensions(BATCH_SIZE, OUTPUT_SIZE));

	Tensor *A = new Tensor(false, Dimensions(BATCH_SIZE, INPUT_SIZE));
	Tensor *B = new Tensor(false, Dimensions(INPUT_SIZE, OUTPUT_SIZE));

	A->rands(0.0f, 1.0f);
	B->rands(0.0f, 1.0f);

	// set_vals(A->get_data(), B->get_data());

	// A->print();
	// B->print();

	{
		C1->zeros();

		sw.start();

		matmul(A->get_data(), B->get_data(), C1->get_data());

		printf("\n");

		sw.stop();
		sw.print_elapsed_seconds();
	}

	{
		C2->zeros();

		A->to_cuda();
		B->to_cuda();

		sw.start();

		k_matmul_1<<<((BATCH_SIZE * OUTPUT_SIZE) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->get_data(), B->get_data(), C2->get_data());

		printf("\n");

		sw.stop();
		sw.print_elapsed_seconds();
	}

	{
		C3->zeros();

		A->to_cuda();
		B->to_cuda();

		sw.start();

		k_matmul_2<<<((BATCH_SIZE * INPUT_SIZE * OUTPUT_SIZE) / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(A->get_data(), B->get_data(), C3->get_data());

		printf("\n");

		sw.stop();
		sw.print_elapsed_seconds();
	}

	printf("\n");

	C2->to_cpu();
	C3->to_cpu();

	C1->print();
	C2->print();
	C3->print();

	delete C1;
	delete C2;
	delete C3;

	delete A;
	delete B;

	return 0;
}