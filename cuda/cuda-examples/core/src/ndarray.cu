#include "hip/hip_runtime.h"
#include "ndarray.cuh"

__global__ void k_set_all(float *data, int cnt, float val)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt)
    {
        data[tid] = val;
    }
}

Shape::Shape()
{
}

Shape::Shape(int dim_1)
{
    this->dims_.push_back(dim_1);
}

Shape::Shape(int dim_1, int dim_2)
{
    this->dims_.push_back(dim_1);
    this->dims_.push_back(dim_2);
}

Shape::Shape(int dim_1, int dim_2, int dim_3)
{
    this->dims_.push_back(dim_1);
    this->dims_.push_back(dim_2);
    this->dims_.push_back(dim_3);
}

Shape::Shape(int dim_1, int dim_2, int dim_3, int dim_4)
{
    this->dims_.push_back(dim_1);
    this->dims_.push_back(dim_2);
    this->dims_.push_back(dim_3);
    this->dims_.push_back(dim_4);
}

Shape::Shape(std::vector<int> dims)
{
    this->dims_ = dims;
}

Shape::~Shape()
{
}

int Shape::operator[](int idx) const
{
    return this->dims_[idx];
}

void Shape::print()
{
    int cnt = this->num_dims();

    for (int i = 0; i < cnt; i++)
    {
        printf("%d", this->dims_[i]);

        if (i < cnt - 1)
        {
            printf("x");
        }
    }

    printf("\n");
}

std::vector<int> Shape::dims()
{
    return this->dims_;
}

int Shape::num_dims()
{
    return this->dims_.size();
}

int Shape::dims_size()
{
    int size = 1;

    for (int i = 0; i < this->num_dims(); i++)
    {
        size *= this->dims_[i];
    }

    return size;
}

NdArray::NdArray(NdArray &src)
{
    this->cuda_ = src.cuda_;
    this->shape_ = src.shape_;

    size_t size = this->size();

    if (src.cuda_)
    {
        hipMalloc(&this->data_, size);
        hipMemcpy(this->data_, src.data_, size, hipMemcpyDeviceToDevice);
    }
    else
    {
        this->data_ = (float *)malloc(size);
        memcpy(this->data_, src.data_, size);
    }
}

NdArray::NdArray(bool cuda, Shape shape)
{
    this->cuda_ = cuda;
    this->shape_ = shape;

    size_t size = this->size();

    if (cuda)
    {
        hipMalloc(&this->data_, size);
    }
    else
    {
        this->data_ = (float *)malloc(size);
    }
}

NdArray::~NdArray()
{
    if (this->cuda_)
    {
        hipFree(this->data_);
    }
    else
    {
        free(this->data_);
    }
}

NdArray *NdArray::zeros(bool cuda, Shape shape)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->zeros();

    return arr;
}

NdArray *NdArray::ones(bool cuda, Shape shape)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->ones();

    return arr;
}

NdArray *NdArray::rands(bool cuda, Shape shape, float mean, float stddev)
{
    NdArray *arr = new NdArray(cuda, shape);

    arr->rands(mean, stddev);

    return arr;
}

void NdArray::print()
{
    bool orig_cuda = this->cuda_;
    this->to_cpu();

    switch (this->num_dims())
    {
    case 1:
    {
        int cnt = this->count();
        printf("[ ");
        for (int i = 0; i < cnt; i++)
        {
            float val = this->data_[i];

            if (i == cnt - 1)
            {
                if (val >= 0.0f)
                {
                    printf(" %f", val);
                }
                else
                {
                    printf("%f", val);
                }
            }
            else
            {
                if (val >= 0.0f)
                {
                    printf(" %f\t", val);
                }
                else
                {
                    printf("%f\t", val);
                }
            }
        }
        printf(" ]");
    }
    break;
    case 2:
    {
        int row_cnt = this->shape_[0];
        int col_cnt = this->shape_[1];

        printf("[");
        for (int i = 0; i < row_cnt; i++)
        {
            if (i == 0)
            {
                printf(" [ ");
            }
            else
            {
                printf("  [ ");
            }

            for (int j = 0; j < col_cnt; j++)
            {
                float val = this->data_[i * col_cnt + j];

                if (j == col_cnt - 1)
                {
                    if (val >= 0.0f)
                    {
                        printf(" %f", val);
                    }
                    else
                    {
                        printf("%f", val);
                    }
                }
                else
                {
                    if (val >= 0.0f)
                    {
                        printf(" %f\t", val);
                    }
                    else
                    {
                        printf("%f\t", val);
                    }
                }
            }

            if (i == row_cnt - 1)
            {
                printf(" ] ");
            }
            else
            {
                printf(" ],\n");
            }
        }
        printf("]\n");
    }
    break;
    case 3:
    {

        int x_cnt = this->shape_[0];
        int y_cnt = this->shape_[1];
        int z_cnt = this->shape_[2];

        printf("[");
        for (int i = 0; i < x_cnt; i++)
        {
            if (i == 0)
            {
                printf(" [ ");
            }
            else
            {
                printf("  [ ");
            }

            for (int j = 0; j < y_cnt; j++)
            {

                if (j == 0)
                {
                    printf(" [ ");
                }
                else
                {
                    printf("  [ ");
                }

                for (int k = 0; k < z_cnt; k++)
                {
                    float val = this->data_[(i * y_cnt * z_cnt) + (j * z_cnt) + k];

                    if (k == z_cnt - 1)
                    {
                        if (val >= 0.0f)
                        {
                            printf(" %f", val);
                        }
                        else
                        {
                            printf("%f", val);
                        }
                    }
                    else
                    {
                        if (val >= 0.0f)
                        {
                            printf(" %f\t", val);
                        }
                        else
                        {
                            printf("%f\t", val);
                        }
                    }
                }

                if (j == y_cnt - 1)
                {
                    printf(" ] ");
                }
                else
                {
                    printf(" ],\n");
                }
            }

            if (i == x_cnt - 1)
            {
                printf(" ] ");
            }
            else
            {
                printf(" ],\n");
            }
        }
        printf("]\n");
    }
    break;
    default:
    {
        printf("Shape: ");
        this->shape_.print();

        printf("Data: \n");
        for (int i = 0; i < this->shape_.dims_size(); i++)
        {
            printf("%d: %f\n", i, this->data_[i]);
        }
    }
    break;
    }

    if (orig_cuda)
    {
        this->to_cuda();
    }
}

void NdArray::copy(NdArray *src)
{
    this->cuda_ = src->cuda_;
    this->shape_ = src->shape_;
    hipMemcpy(this->data_, src->data_, src->size(), hipMemcpyDefault);
}

void NdArray::reshape(Shape shape)
{
    this->shape_ = shape;

    if (this->cuda_)
    {
        hipFree(this->data_);
        hipMalloc(&this->data_, this->size());
    }
    else
    {
        free(this->data_);
        this->data_ = (float *)malloc(this->size());
    }
}

void NdArray::change_dim(int dim_idx, int dim)
{
    std::vector<int> dims = this->shape_.dims();
    dims[dim_idx] = dim;
    this->reshape(Shape(dims));
}

bool NdArray::is_cuda()
{
    return this->cuda_;
}

void NdArray::to_cpu()
{
    if (this->cuda_)
    {
        size_t size = this->size();
        float *dst = (float *)malloc(size);
        hipMemcpy(dst, this->data_, size, hipMemcpyDeviceToHost);
        hipFree(this->data_);
        this->data_ = dst;
        this->cuda_ = false;
    }
}

void NdArray::to_cuda()
{
    if (!this->cuda_)
    {
        size_t size = this->size();
        float *dst;
        hipMalloc(&dst, size);
        hipMemcpy(dst, this->data_, size, hipMemcpyHostToDevice);
        free(this->data_);
        this->data_ = dst;
        this->cuda_ = true;
    }
}

Shape NdArray::shape()
{
    return this->shape_;
}

int NdArray::num_dims()
{
    return this->shape_.num_dims();
}

int NdArray::dims_size()
{
    return this->shape_.dims_size();
}

int NdArray::count()
{
    return this->shape_.dims_size();
}

size_t NdArray::size()
{
    return sizeof(float) * this->shape_.dims_size();
}

float *NdArray::data()
{
    return this->data_;
}

void NdArray::zeros()
{
    size_t size = this->size();

    if (this->cuda_)
    {
        hipMemset(this->data_, 0, size);
    }
    else
    {
        memset(this->data_, 0, size);
    }
}

void NdArray::ones()
{
    if (this->is_cuda())
    {
        k_set_all<<<(this->count() / THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(this->data_, this->count(), 1.0f);
    }
    else
    {
        for (int i = 0; i < this->count(); i++)
        {
            this->data_[i] = 1.0f;
        }
    }
}

void NdArray::rands(float mean, float stddev)
{
    bool orig_cuda = this->cuda_;

    this->to_cpu();

    {
        std::random_device rd;
        std::mt19937 gen(rd());

        for (int i = 0; i < this->shape_.dims_size(); i++)
        {
            std::normal_distribution<float> d(mean, stddev);
            this->data_[i] = d(gen);
        }
    }

    if (orig_cuda)
    {
        this->to_cuda();
    }
}

float NdArray::get_val(int idx)
{
    float val;
    hipMemcpy(&val, &this->data_[idx], sizeof(float), hipMemcpyDefault);
    return val;
}

void NdArray::set_val(int idx, float val)
{
    hipMemcpy(&this->data_[idx], &val, sizeof(float), hipMemcpyDefault);
}

float NdArray::sum()
{
    float sum = 0.0f;

    for (int i = 0; i < this->count(); i++)
    {
        sum += this->get_val(i);
    }

    return sum;
}

float NdArray::min()
{
    float min = FLT_MAX;

    float val = 0;

    for (int i = 0; i < this->count(); i++)
    {
        val = this->get_val(i);

        if (val < min)
        {
            min = val;
        }
    }

    return min;
}

float NdArray::max()
{
    float max = -FLT_MAX;

    float val = 0;

    for (int i = 0; i < this->count(); i++)
    {
        val = this->get_val(i);

        if (val > max)
        {
            max = val;
        }
    }

    return max;
}

float NdArray::mean()
{
    return this->sum() / this->count();
}

float NdArray::stddev()
{
    float stddev = 0.0f;

    float mean = this->mean();

    for (int i = 0; i < this->count(); i++)
    {
        float diff = this->get_val(i) - mean;
        stddev == diff *diff;
    }

    stddev /= this->count();

    return sqrt(stddev);
}